
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(64) main_kernel0(float* __restrict__ inputs, float* __restrict__ weight, float* __restrict__ conv2d_nhwc) {
  float conv2d_nhwc_local[56];
  __shared__ float PadInput_shared[1792];
  __shared__ float weight_shared[2048];
  conv2d_nhwc_local[0] = 0.000000e+00f;
  conv2d_nhwc_local[7] = 0.000000e+00f;
  conv2d_nhwc_local[14] = 0.000000e+00f;
  conv2d_nhwc_local[21] = 0.000000e+00f;
  conv2d_nhwc_local[28] = 0.000000e+00f;
  conv2d_nhwc_local[35] = 0.000000e+00f;
  conv2d_nhwc_local[42] = 0.000000e+00f;
  conv2d_nhwc_local[49] = 0.000000e+00f;
  conv2d_nhwc_local[1] = 0.000000e+00f;
  conv2d_nhwc_local[8] = 0.000000e+00f;
  conv2d_nhwc_local[15] = 0.000000e+00f;
  conv2d_nhwc_local[22] = 0.000000e+00f;
  conv2d_nhwc_local[29] = 0.000000e+00f;
  conv2d_nhwc_local[36] = 0.000000e+00f;
  conv2d_nhwc_local[43] = 0.000000e+00f;
  conv2d_nhwc_local[50] = 0.000000e+00f;
  conv2d_nhwc_local[2] = 0.000000e+00f;
  conv2d_nhwc_local[9] = 0.000000e+00f;
  conv2d_nhwc_local[16] = 0.000000e+00f;
  conv2d_nhwc_local[23] = 0.000000e+00f;
  conv2d_nhwc_local[30] = 0.000000e+00f;
  conv2d_nhwc_local[37] = 0.000000e+00f;
  conv2d_nhwc_local[44] = 0.000000e+00f;
  conv2d_nhwc_local[51] = 0.000000e+00f;
  conv2d_nhwc_local[3] = 0.000000e+00f;
  conv2d_nhwc_local[10] = 0.000000e+00f;
  conv2d_nhwc_local[17] = 0.000000e+00f;
  conv2d_nhwc_local[24] = 0.000000e+00f;
  conv2d_nhwc_local[31] = 0.000000e+00f;
  conv2d_nhwc_local[38] = 0.000000e+00f;
  conv2d_nhwc_local[45] = 0.000000e+00f;
  conv2d_nhwc_local[52] = 0.000000e+00f;
  conv2d_nhwc_local[4] = 0.000000e+00f;
  conv2d_nhwc_local[11] = 0.000000e+00f;
  conv2d_nhwc_local[18] = 0.000000e+00f;
  conv2d_nhwc_local[25] = 0.000000e+00f;
  conv2d_nhwc_local[32] = 0.000000e+00f;
  conv2d_nhwc_local[39] = 0.000000e+00f;
  conv2d_nhwc_local[46] = 0.000000e+00f;
  conv2d_nhwc_local[53] = 0.000000e+00f;
  conv2d_nhwc_local[5] = 0.000000e+00f;
  conv2d_nhwc_local[12] = 0.000000e+00f;
  conv2d_nhwc_local[19] = 0.000000e+00f;
  conv2d_nhwc_local[26] = 0.000000e+00f;
  conv2d_nhwc_local[33] = 0.000000e+00f;
  conv2d_nhwc_local[40] = 0.000000e+00f;
  conv2d_nhwc_local[47] = 0.000000e+00f;
  conv2d_nhwc_local[54] = 0.000000e+00f;
  conv2d_nhwc_local[6] = 0.000000e+00f;
  conv2d_nhwc_local[13] = 0.000000e+00f;
  conv2d_nhwc_local[20] = 0.000000e+00f;
  conv2d_nhwc_local[27] = 0.000000e+00f;
  conv2d_nhwc_local[34] = 0.000000e+00f;
  conv2d_nhwc_local[41] = 0.000000e+00f;
  conv2d_nhwc_local[48] = 0.000000e+00f;
  conv2d_nhwc_local[55] = 0.000000e+00f;
  for (int rc_0 = 0; rc_0 < 2; ++rc_0) {
    __syncthreads();
    *(float2*)(PadInput_shared + (((int)threadIdx.x) * 2)) = *(float2*)(inputs + (((((((((int)blockIdx.x) / 112) * 100352) + ((((int)threadIdx.x) >> 5) * 3584)) + (((((int)blockIdx.x) % 112) >> 2) * 128)) + (((((int)threadIdx.x) & 31) >> 4) * 64)) + (rc_0 * 32)) + ((((int)threadIdx.x) & 15) * 2)));
    *(float2*)(PadInput_shared + ((((int)threadIdx.x) * 2) + 128)) = *(float2*)(inputs + ((((((((((int)blockIdx.x) / 112) * 100352) + ((((int)threadIdx.x) >> 5) * 3584)) + (((((int)blockIdx.x) % 112) >> 2) * 128)) + (((((int)threadIdx.x) & 31) >> 4) * 64)) + (rc_0 * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 7168));
    *(float2*)(PadInput_shared + ((((int)threadIdx.x) * 2) + 256)) = *(float2*)(inputs + ((((((((((int)blockIdx.x) / 112) * 100352) + ((((int)threadIdx.x) >> 5) * 3584)) + (((((int)blockIdx.x) % 112) >> 2) * 128)) + (((((int)threadIdx.x) & 31) >> 4) * 64)) + (rc_0 * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 14336));
    *(float2*)(PadInput_shared + ((((int)threadIdx.x) * 2) + 384)) = *(float2*)(inputs + ((((((((((int)blockIdx.x) / 112) * 100352) + ((((int)threadIdx.x) >> 5) * 3584)) + (((((int)blockIdx.x) % 112) >> 2) * 128)) + (((((int)threadIdx.x) & 31) >> 4) * 64)) + (rc_0 * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 21504));
    *(float2*)(PadInput_shared + ((((int)threadIdx.x) * 2) + 512)) = *(float2*)(inputs + ((((((((((int)blockIdx.x) / 112) * 100352) + ((((int)threadIdx.x) >> 5) * 3584)) + (((((int)blockIdx.x) % 112) >> 2) * 128)) + (((((int)threadIdx.x) & 31) >> 4) * 64)) + (rc_0 * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 28672));
    *(float2*)(PadInput_shared + ((((int)threadIdx.x) * 2) + 640)) = *(float2*)(inputs + ((((((((((int)blockIdx.x) / 112) * 100352) + ((((int)threadIdx.x) >> 5) * 3584)) + (((((int)blockIdx.x) % 112) >> 2) * 128)) + (((((int)threadIdx.x) & 31) >> 4) * 64)) + (rc_0 * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 35840));
    *(float2*)(PadInput_shared + ((((int)threadIdx.x) * 2) + 768)) = *(float2*)(inputs + ((((((((((int)blockIdx.x) / 112) * 100352) + ((((int)threadIdx.x) >> 5) * 3584)) + (((((int)blockIdx.x) % 112) >> 2) * 128)) + (((((int)threadIdx.x) & 31) >> 4) * 64)) + (rc_0 * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 43008));
    *(float2*)(PadInput_shared + ((((int)threadIdx.x) * 2) + 896)) = *(float2*)(inputs + ((((((((((int)blockIdx.x) / 112) * 100352) + ((((int)threadIdx.x) >> 5) * 3584)) + (((((int)blockIdx.x) % 112) >> 2) * 128)) + (((((int)threadIdx.x) & 31) >> 4) * 64)) + (rc_0 * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 50176));
    *(float2*)(PadInput_shared + ((((int)threadIdx.x) * 2) + 1024)) = *(float2*)(inputs + ((((((((((int)blockIdx.x) / 112) * 100352) + ((((int)threadIdx.x) >> 5) * 3584)) + (((((int)blockIdx.x) % 112) >> 2) * 128)) + (((((int)threadIdx.x) & 31) >> 4) * 64)) + (rc_0 * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 57344));
    *(float2*)(PadInput_shared + ((((int)threadIdx.x) * 2) + 1152)) = *(float2*)(inputs + ((((((((((int)blockIdx.x) / 112) * 100352) + ((((int)threadIdx.x) >> 5) * 3584)) + (((((int)blockIdx.x) % 112) >> 2) * 128)) + (((((int)threadIdx.x) & 31) >> 4) * 64)) + (rc_0 * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 64512));
    *(float2*)(PadInput_shared + ((((int)threadIdx.x) * 2) + 1280)) = *(float2*)(inputs + ((((((((((int)blockIdx.x) / 112) * 100352) + ((((int)threadIdx.x) >> 5) * 3584)) + (((((int)blockIdx.x) % 112) >> 2) * 128)) + (((((int)threadIdx.x) & 31) >> 4) * 64)) + (rc_0 * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 71680));
    *(float2*)(PadInput_shared + ((((int)threadIdx.x) * 2) + 1408)) = *(float2*)(inputs + ((((((((((int)blockIdx.x) / 112) * 100352) + ((((int)threadIdx.x) >> 5) * 3584)) + (((((int)blockIdx.x) % 112) >> 2) * 128)) + (((((int)threadIdx.x) & 31) >> 4) * 64)) + (rc_0 * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 78848));
    *(float2*)(PadInput_shared + ((((int)threadIdx.x) * 2) + 1536)) = *(float2*)(inputs + ((((((((((int)blockIdx.x) / 112) * 100352) + ((((int)threadIdx.x) >> 5) * 3584)) + (((((int)blockIdx.x) % 112) >> 2) * 128)) + (((((int)threadIdx.x) & 31) >> 4) * 64)) + (rc_0 * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 86016));
    *(float2*)(PadInput_shared + ((((int)threadIdx.x) * 2) + 1664)) = *(float2*)(inputs + ((((((((((int)blockIdx.x) / 112) * 100352) + ((((int)threadIdx.x) >> 5) * 3584)) + (((((int)blockIdx.x) % 112) >> 2) * 128)) + (((((int)threadIdx.x) & 31) >> 4) * 64)) + (rc_0 * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 93184));
    *(float4*)(weight_shared + (((int)threadIdx.x) * 4)) = *(float4*)(weight + ((((rc_0 * 8192) + ((((int)threadIdx.x) >> 4) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) & 15) * 4)));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 256)) = *(float4*)(weight + (((((rc_0 * 8192) + ((((int)threadIdx.x) >> 4) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 1024));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 512)) = *(float4*)(weight + (((((rc_0 * 8192) + ((((int)threadIdx.x) >> 4) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 2048));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 768)) = *(float4*)(weight + (((((rc_0 * 8192) + ((((int)threadIdx.x) >> 4) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 3072));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 1024)) = *(float4*)(weight + (((((rc_0 * 8192) + ((((int)threadIdx.x) >> 4) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 4096));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 1280)) = *(float4*)(weight + (((((rc_0 * 8192) + ((((int)threadIdx.x) >> 4) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 5120));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 1536)) = *(float4*)(weight + (((((rc_0 * 8192) + ((((int)threadIdx.x) >> 4) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 6144));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 1792)) = *(float4*)(weight + (((((rc_0 * 8192) + ((((int)threadIdx.x) >> 4) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 7168));
    __syncthreads();
    for (int rc_1 = 0; rc_1 < 32; ++rc_1) {
      conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((((int)threadIdx.x) >> 5) * 448) + (((((int)threadIdx.x) & 31) >> 4) * 32)) + rc_1)] * weight_shared[((rc_1 * 64) + (((int)threadIdx.x) & 15))]));
      conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((((int)threadIdx.x) >> 5) * 448) + (((((int)threadIdx.x) & 31) >> 4) * 32)) + rc_1)] * weight_shared[(((rc_1 * 64) + (((int)threadIdx.x) & 15)) + 16)]));
      conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((((int)threadIdx.x) >> 5) * 448) + (((((int)threadIdx.x) & 31) >> 4) * 32)) + rc_1)] * weight_shared[(((rc_1 * 64) + (((int)threadIdx.x) & 15)) + 32)]));
      conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((((int)threadIdx.x) >> 5) * 448) + (((((int)threadIdx.x) & 31) >> 4) * 32)) + rc_1)] * weight_shared[(((rc_1 * 64) + (((int)threadIdx.x) & 15)) + 48)]));
      conv2d_nhwc_local[28] = (conv2d_nhwc_local[28] + (PadInput_shared[(((((((int)threadIdx.x) >> 5) * 448) + (((((int)threadIdx.x) & 31) >> 4) * 32)) + rc_1) + 896)] * weight_shared[((rc_1 * 64) + (((int)threadIdx.x) & 15))]));
      conv2d_nhwc_local[35] = (conv2d_nhwc_local[35] + (PadInput_shared[(((((((int)threadIdx.x) >> 5) * 448) + (((((int)threadIdx.x) & 31) >> 4) * 32)) + rc_1) + 896)] * weight_shared[(((rc_1 * 64) + (((int)threadIdx.x) & 15)) + 16)]));
      conv2d_nhwc_local[42] = (conv2d_nhwc_local[42] + (PadInput_shared[(((((((int)threadIdx.x) >> 5) * 448) + (((((int)threadIdx.x) & 31) >> 4) * 32)) + rc_1) + 896)] * weight_shared[(((rc_1 * 64) + (((int)threadIdx.x) & 15)) + 32)]));
      conv2d_nhwc_local[49] = (conv2d_nhwc_local[49] + (PadInput_shared[(((((((int)threadIdx.x) >> 5) * 448) + (((((int)threadIdx.x) & 31) >> 4) * 32)) + rc_1) + 896)] * weight_shared[(((rc_1 * 64) + (((int)threadIdx.x) & 15)) + 48)]));
      conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((((int)threadIdx.x) >> 5) * 448) + (((((int)threadIdx.x) & 31) >> 4) * 32)) + rc_1) + 64)] * weight_shared[((rc_1 * 64) + (((int)threadIdx.x) & 15))]));
      conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((((int)threadIdx.x) >> 5) * 448) + (((((int)threadIdx.x) & 31) >> 4) * 32)) + rc_1) + 64)] * weight_shared[(((rc_1 * 64) + (((int)threadIdx.x) & 15)) + 16)]));
      conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((((int)threadIdx.x) >> 5) * 448) + (((((int)threadIdx.x) & 31) >> 4) * 32)) + rc_1) + 64)] * weight_shared[(((rc_1 * 64) + (((int)threadIdx.x) & 15)) + 32)]));
      conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((((int)threadIdx.x) >> 5) * 448) + (((((int)threadIdx.x) & 31) >> 4) * 32)) + rc_1) + 64)] * weight_shared[(((rc_1 * 64) + (((int)threadIdx.x) & 15)) + 48)]));
      conv2d_nhwc_local[29] = (conv2d_nhwc_local[29] + (PadInput_shared[(((((((int)threadIdx.x) >> 5) * 448) + (((((int)threadIdx.x) & 31) >> 4) * 32)) + rc_1) + 960)] * weight_shared[((rc_1 * 64) + (((int)threadIdx.x) & 15))]));
      conv2d_nhwc_local[36] = (conv2d_nhwc_local[36] + (PadInput_shared[(((((((int)threadIdx.x) >> 5) * 448) + (((((int)threadIdx.x) & 31) >> 4) * 32)) + rc_1) + 960)] * weight_shared[(((rc_1 * 64) + (((int)threadIdx.x) & 15)) + 16)]));
      conv2d_nhwc_local[43] = (conv2d_nhwc_local[43] + (PadInput_shared[(((((((int)threadIdx.x) >> 5) * 448) + (((((int)threadIdx.x) & 31) >> 4) * 32)) + rc_1) + 960)] * weight_shared[(((rc_1 * 64) + (((int)threadIdx.x) & 15)) + 32)]));
      conv2d_nhwc_local[50] = (conv2d_nhwc_local[50] + (PadInput_shared[(((((((int)threadIdx.x) >> 5) * 448) + (((((int)threadIdx.x) & 31) >> 4) * 32)) + rc_1) + 960)] * weight_shared[(((rc_1 * 64) + (((int)threadIdx.x) & 15)) + 48)]));
      conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((((int)threadIdx.x) >> 5) * 448) + (((((int)threadIdx.x) & 31) >> 4) * 32)) + rc_1) + 128)] * weight_shared[((rc_1 * 64) + (((int)threadIdx.x) & 15))]));
      conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((((int)threadIdx.x) >> 5) * 448) + (((((int)threadIdx.x) & 31) >> 4) * 32)) + rc_1) + 128)] * weight_shared[(((rc_1 * 64) + (((int)threadIdx.x) & 15)) + 16)]));
      conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((((int)threadIdx.x) >> 5) * 448) + (((((int)threadIdx.x) & 31) >> 4) * 32)) + rc_1) + 128)] * weight_shared[(((rc_1 * 64) + (((int)threadIdx.x) & 15)) + 32)]));
      conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((((int)threadIdx.x) >> 5) * 448) + (((((int)threadIdx.x) & 31) >> 4) * 32)) + rc_1) + 128)] * weight_shared[(((rc_1 * 64) + (((int)threadIdx.x) & 15)) + 48)]));
      conv2d_nhwc_local[30] = (conv2d_nhwc_local[30] + (PadInput_shared[(((((((int)threadIdx.x) >> 5) * 448) + (((((int)threadIdx.x) & 31) >> 4) * 32)) + rc_1) + 1024)] * weight_shared[((rc_1 * 64) + (((int)threadIdx.x) & 15))]));
      conv2d_nhwc_local[37] = (conv2d_nhwc_local[37] + (PadInput_shared[(((((((int)threadIdx.x) >> 5) * 448) + (((((int)threadIdx.x) & 31) >> 4) * 32)) + rc_1) + 1024)] * weight_shared[(((rc_1 * 64) + (((int)threadIdx.x) & 15)) + 16)]));
      conv2d_nhwc_local[44] = (conv2d_nhwc_local[44] + (PadInput_shared[(((((((int)threadIdx.x) >> 5) * 448) + (((((int)threadIdx.x) & 31) >> 4) * 32)) + rc_1) + 1024)] * weight_shared[(((rc_1 * 64) + (((int)threadIdx.x) & 15)) + 32)]));
      conv2d_nhwc_local[51] = (conv2d_nhwc_local[51] + (PadInput_shared[(((((((int)threadIdx.x) >> 5) * 448) + (((((int)threadIdx.x) & 31) >> 4) * 32)) + rc_1) + 1024)] * weight_shared[(((rc_1 * 64) + (((int)threadIdx.x) & 15)) + 48)]));
      conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((((int)threadIdx.x) >> 5) * 448) + (((((int)threadIdx.x) & 31) >> 4) * 32)) + rc_1) + 192)] * weight_shared[((rc_1 * 64) + (((int)threadIdx.x) & 15))]));
      conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((((int)threadIdx.x) >> 5) * 448) + (((((int)threadIdx.x) & 31) >> 4) * 32)) + rc_1) + 192)] * weight_shared[(((rc_1 * 64) + (((int)threadIdx.x) & 15)) + 16)]));
      conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((((int)threadIdx.x) >> 5) * 448) + (((((int)threadIdx.x) & 31) >> 4) * 32)) + rc_1) + 192)] * weight_shared[(((rc_1 * 64) + (((int)threadIdx.x) & 15)) + 32)]));
      conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((((int)threadIdx.x) >> 5) * 448) + (((((int)threadIdx.x) & 31) >> 4) * 32)) + rc_1) + 192)] * weight_shared[(((rc_1 * 64) + (((int)threadIdx.x) & 15)) + 48)]));
      conv2d_nhwc_local[31] = (conv2d_nhwc_local[31] + (PadInput_shared[(((((((int)threadIdx.x) >> 5) * 448) + (((((int)threadIdx.x) & 31) >> 4) * 32)) + rc_1) + 1088)] * weight_shared[((rc_1 * 64) + (((int)threadIdx.x) & 15))]));
      conv2d_nhwc_local[38] = (conv2d_nhwc_local[38] + (PadInput_shared[(((((((int)threadIdx.x) >> 5) * 448) + (((((int)threadIdx.x) & 31) >> 4) * 32)) + rc_1) + 1088)] * weight_shared[(((rc_1 * 64) + (((int)threadIdx.x) & 15)) + 16)]));
      conv2d_nhwc_local[45] = (conv2d_nhwc_local[45] + (PadInput_shared[(((((((int)threadIdx.x) >> 5) * 448) + (((((int)threadIdx.x) & 31) >> 4) * 32)) + rc_1) + 1088)] * weight_shared[(((rc_1 * 64) + (((int)threadIdx.x) & 15)) + 32)]));
      conv2d_nhwc_local[52] = (conv2d_nhwc_local[52] + (PadInput_shared[(((((((int)threadIdx.x) >> 5) * 448) + (((((int)threadIdx.x) & 31) >> 4) * 32)) + rc_1) + 1088)] * weight_shared[(((rc_1 * 64) + (((int)threadIdx.x) & 15)) + 48)]));
      conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((((int)threadIdx.x) >> 5) * 448) + (((((int)threadIdx.x) & 31) >> 4) * 32)) + rc_1) + 256)] * weight_shared[((rc_1 * 64) + (((int)threadIdx.x) & 15))]));
      conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((((int)threadIdx.x) >> 5) * 448) + (((((int)threadIdx.x) & 31) >> 4) * 32)) + rc_1) + 256)] * weight_shared[(((rc_1 * 64) + (((int)threadIdx.x) & 15)) + 16)]));
      conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((((int)threadIdx.x) >> 5) * 448) + (((((int)threadIdx.x) & 31) >> 4) * 32)) + rc_1) + 256)] * weight_shared[(((rc_1 * 64) + (((int)threadIdx.x) & 15)) + 32)]));
      conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((((int)threadIdx.x) >> 5) * 448) + (((((int)threadIdx.x) & 31) >> 4) * 32)) + rc_1) + 256)] * weight_shared[(((rc_1 * 64) + (((int)threadIdx.x) & 15)) + 48)]));
      conv2d_nhwc_local[32] = (conv2d_nhwc_local[32] + (PadInput_shared[(((((((int)threadIdx.x) >> 5) * 448) + (((((int)threadIdx.x) & 31) >> 4) * 32)) + rc_1) + 1152)] * weight_shared[((rc_1 * 64) + (((int)threadIdx.x) & 15))]));
      conv2d_nhwc_local[39] = (conv2d_nhwc_local[39] + (PadInput_shared[(((((((int)threadIdx.x) >> 5) * 448) + (((((int)threadIdx.x) & 31) >> 4) * 32)) + rc_1) + 1152)] * weight_shared[(((rc_1 * 64) + (((int)threadIdx.x) & 15)) + 16)]));
      conv2d_nhwc_local[46] = (conv2d_nhwc_local[46] + (PadInput_shared[(((((((int)threadIdx.x) >> 5) * 448) + (((((int)threadIdx.x) & 31) >> 4) * 32)) + rc_1) + 1152)] * weight_shared[(((rc_1 * 64) + (((int)threadIdx.x) & 15)) + 32)]));
      conv2d_nhwc_local[53] = (conv2d_nhwc_local[53] + (PadInput_shared[(((((((int)threadIdx.x) >> 5) * 448) + (((((int)threadIdx.x) & 31) >> 4) * 32)) + rc_1) + 1152)] * weight_shared[(((rc_1 * 64) + (((int)threadIdx.x) & 15)) + 48)]));
      conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((((int)threadIdx.x) >> 5) * 448) + (((((int)threadIdx.x) & 31) >> 4) * 32)) + rc_1) + 320)] * weight_shared[((rc_1 * 64) + (((int)threadIdx.x) & 15))]));
      conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((((int)threadIdx.x) >> 5) * 448) + (((((int)threadIdx.x) & 31) >> 4) * 32)) + rc_1) + 320)] * weight_shared[(((rc_1 * 64) + (((int)threadIdx.x) & 15)) + 16)]));
      conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((((int)threadIdx.x) >> 5) * 448) + (((((int)threadIdx.x) & 31) >> 4) * 32)) + rc_1) + 320)] * weight_shared[(((rc_1 * 64) + (((int)threadIdx.x) & 15)) + 32)]));
      conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((((int)threadIdx.x) >> 5) * 448) + (((((int)threadIdx.x) & 31) >> 4) * 32)) + rc_1) + 320)] * weight_shared[(((rc_1 * 64) + (((int)threadIdx.x) & 15)) + 48)]));
      conv2d_nhwc_local[33] = (conv2d_nhwc_local[33] + (PadInput_shared[(((((((int)threadIdx.x) >> 5) * 448) + (((((int)threadIdx.x) & 31) >> 4) * 32)) + rc_1) + 1216)] * weight_shared[((rc_1 * 64) + (((int)threadIdx.x) & 15))]));
      conv2d_nhwc_local[40] = (conv2d_nhwc_local[40] + (PadInput_shared[(((((((int)threadIdx.x) >> 5) * 448) + (((((int)threadIdx.x) & 31) >> 4) * 32)) + rc_1) + 1216)] * weight_shared[(((rc_1 * 64) + (((int)threadIdx.x) & 15)) + 16)]));
      conv2d_nhwc_local[47] = (conv2d_nhwc_local[47] + (PadInput_shared[(((((((int)threadIdx.x) >> 5) * 448) + (((((int)threadIdx.x) & 31) >> 4) * 32)) + rc_1) + 1216)] * weight_shared[(((rc_1 * 64) + (((int)threadIdx.x) & 15)) + 32)]));
      conv2d_nhwc_local[54] = (conv2d_nhwc_local[54] + (PadInput_shared[(((((((int)threadIdx.x) >> 5) * 448) + (((((int)threadIdx.x) & 31) >> 4) * 32)) + rc_1) + 1216)] * weight_shared[(((rc_1 * 64) + (((int)threadIdx.x) & 15)) + 48)]));
      conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((((int)threadIdx.x) >> 5) * 448) + (((((int)threadIdx.x) & 31) >> 4) * 32)) + rc_1) + 384)] * weight_shared[((rc_1 * 64) + (((int)threadIdx.x) & 15))]));
      conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((((int)threadIdx.x) >> 5) * 448) + (((((int)threadIdx.x) & 31) >> 4) * 32)) + rc_1) + 384)] * weight_shared[(((rc_1 * 64) + (((int)threadIdx.x) & 15)) + 16)]));
      conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((((int)threadIdx.x) >> 5) * 448) + (((((int)threadIdx.x) & 31) >> 4) * 32)) + rc_1) + 384)] * weight_shared[(((rc_1 * 64) + (((int)threadIdx.x) & 15)) + 32)]));
      conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((((int)threadIdx.x) >> 5) * 448) + (((((int)threadIdx.x) & 31) >> 4) * 32)) + rc_1) + 384)] * weight_shared[(((rc_1 * 64) + (((int)threadIdx.x) & 15)) + 48)]));
      conv2d_nhwc_local[34] = (conv2d_nhwc_local[34] + (PadInput_shared[(((((((int)threadIdx.x) >> 5) * 448) + (((((int)threadIdx.x) & 31) >> 4) * 32)) + rc_1) + 1280)] * weight_shared[((rc_1 * 64) + (((int)threadIdx.x) & 15))]));
      conv2d_nhwc_local[41] = (conv2d_nhwc_local[41] + (PadInput_shared[(((((((int)threadIdx.x) >> 5) * 448) + (((((int)threadIdx.x) & 31) >> 4) * 32)) + rc_1) + 1280)] * weight_shared[(((rc_1 * 64) + (((int)threadIdx.x) & 15)) + 16)]));
      conv2d_nhwc_local[48] = (conv2d_nhwc_local[48] + (PadInput_shared[(((((((int)threadIdx.x) >> 5) * 448) + (((((int)threadIdx.x) & 31) >> 4) * 32)) + rc_1) + 1280)] * weight_shared[(((rc_1 * 64) + (((int)threadIdx.x) & 15)) + 32)]));
      conv2d_nhwc_local[55] = (conv2d_nhwc_local[55] + (PadInput_shared[(((((((int)threadIdx.x) >> 5) * 448) + (((((int)threadIdx.x) & 31) >> 4) * 32)) + rc_1) + 1280)] * weight_shared[(((rc_1 * 64) + (((int)threadIdx.x) & 15)) + 48)]));
    }
  }
  conv2d_nhwc[(((((((((int)blockIdx.x) / 112) * 401408) + ((((int)threadIdx.x) >> 5) * 100352)) + (((((int)blockIdx.x) % 112) >> 2) * 512)) + (((((int)threadIdx.x) & 31) >> 4) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((int)threadIdx.x) & 15))] = conv2d_nhwc_local[0];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 112) * 401408) + ((((int)threadIdx.x) >> 5) * 100352)) + (((((int)blockIdx.x) % 112) >> 2) * 512)) + (((((int)threadIdx.x) & 31) >> 4) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((int)threadIdx.x) & 15)) + 16)] = conv2d_nhwc_local[7];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 112) * 401408) + ((((int)threadIdx.x) >> 5) * 100352)) + (((((int)blockIdx.x) % 112) >> 2) * 512)) + (((((int)threadIdx.x) & 31) >> 4) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((int)threadIdx.x) & 15)) + 32)] = conv2d_nhwc_local[14];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 112) * 401408) + ((((int)threadIdx.x) >> 5) * 100352)) + (((((int)blockIdx.x) % 112) >> 2) * 512)) + (((((int)threadIdx.x) & 31) >> 4) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((int)threadIdx.x) & 15)) + 48)] = conv2d_nhwc_local[21];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 112) * 401408) + ((((int)threadIdx.x) >> 5) * 100352)) + (((((int)blockIdx.x) % 112) >> 2) * 512)) + (((((int)threadIdx.x) & 31) >> 4) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((int)threadIdx.x) & 15)) + 200704)] = conv2d_nhwc_local[28];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 112) * 401408) + ((((int)threadIdx.x) >> 5) * 100352)) + (((((int)blockIdx.x) % 112) >> 2) * 512)) + (((((int)threadIdx.x) & 31) >> 4) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((int)threadIdx.x) & 15)) + 200720)] = conv2d_nhwc_local[35];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 112) * 401408) + ((((int)threadIdx.x) >> 5) * 100352)) + (((((int)blockIdx.x) % 112) >> 2) * 512)) + (((((int)threadIdx.x) & 31) >> 4) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((int)threadIdx.x) & 15)) + 200736)] = conv2d_nhwc_local[42];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 112) * 401408) + ((((int)threadIdx.x) >> 5) * 100352)) + (((((int)blockIdx.x) % 112) >> 2) * 512)) + (((((int)threadIdx.x) & 31) >> 4) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((int)threadIdx.x) & 15)) + 200752)] = conv2d_nhwc_local[49];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 112) * 401408) + ((((int)threadIdx.x) >> 5) * 100352)) + (((((int)blockIdx.x) % 112) >> 2) * 512)) + (((((int)threadIdx.x) & 31) >> 4) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((int)threadIdx.x) & 15)) + 14336)] = conv2d_nhwc_local[1];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 112) * 401408) + ((((int)threadIdx.x) >> 5) * 100352)) + (((((int)blockIdx.x) % 112) >> 2) * 512)) + (((((int)threadIdx.x) & 31) >> 4) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((int)threadIdx.x) & 15)) + 14352)] = conv2d_nhwc_local[8];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 112) * 401408) + ((((int)threadIdx.x) >> 5) * 100352)) + (((((int)blockIdx.x) % 112) >> 2) * 512)) + (((((int)threadIdx.x) & 31) >> 4) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((int)threadIdx.x) & 15)) + 14368)] = conv2d_nhwc_local[15];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 112) * 401408) + ((((int)threadIdx.x) >> 5) * 100352)) + (((((int)blockIdx.x) % 112) >> 2) * 512)) + (((((int)threadIdx.x) & 31) >> 4) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((int)threadIdx.x) & 15)) + 14384)] = conv2d_nhwc_local[22];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 112) * 401408) + ((((int)threadIdx.x) >> 5) * 100352)) + (((((int)blockIdx.x) % 112) >> 2) * 512)) + (((((int)threadIdx.x) & 31) >> 4) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((int)threadIdx.x) & 15)) + 215040)] = conv2d_nhwc_local[29];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 112) * 401408) + ((((int)threadIdx.x) >> 5) * 100352)) + (((((int)blockIdx.x) % 112) >> 2) * 512)) + (((((int)threadIdx.x) & 31) >> 4) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((int)threadIdx.x) & 15)) + 215056)] = conv2d_nhwc_local[36];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 112) * 401408) + ((((int)threadIdx.x) >> 5) * 100352)) + (((((int)blockIdx.x) % 112) >> 2) * 512)) + (((((int)threadIdx.x) & 31) >> 4) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((int)threadIdx.x) & 15)) + 215072)] = conv2d_nhwc_local[43];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 112) * 401408) + ((((int)threadIdx.x) >> 5) * 100352)) + (((((int)blockIdx.x) % 112) >> 2) * 512)) + (((((int)threadIdx.x) & 31) >> 4) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((int)threadIdx.x) & 15)) + 215088)] = conv2d_nhwc_local[50];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 112) * 401408) + ((((int)threadIdx.x) >> 5) * 100352)) + (((((int)blockIdx.x) % 112) >> 2) * 512)) + (((((int)threadIdx.x) & 31) >> 4) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((int)threadIdx.x) & 15)) + 28672)] = conv2d_nhwc_local[2];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 112) * 401408) + ((((int)threadIdx.x) >> 5) * 100352)) + (((((int)blockIdx.x) % 112) >> 2) * 512)) + (((((int)threadIdx.x) & 31) >> 4) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((int)threadIdx.x) & 15)) + 28688)] = conv2d_nhwc_local[9];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 112) * 401408) + ((((int)threadIdx.x) >> 5) * 100352)) + (((((int)blockIdx.x) % 112) >> 2) * 512)) + (((((int)threadIdx.x) & 31) >> 4) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((int)threadIdx.x) & 15)) + 28704)] = conv2d_nhwc_local[16];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 112) * 401408) + ((((int)threadIdx.x) >> 5) * 100352)) + (((((int)blockIdx.x) % 112) >> 2) * 512)) + (((((int)threadIdx.x) & 31) >> 4) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((int)threadIdx.x) & 15)) + 28720)] = conv2d_nhwc_local[23];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 112) * 401408) + ((((int)threadIdx.x) >> 5) * 100352)) + (((((int)blockIdx.x) % 112) >> 2) * 512)) + (((((int)threadIdx.x) & 31) >> 4) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((int)threadIdx.x) & 15)) + 229376)] = conv2d_nhwc_local[30];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 112) * 401408) + ((((int)threadIdx.x) >> 5) * 100352)) + (((((int)blockIdx.x) % 112) >> 2) * 512)) + (((((int)threadIdx.x) & 31) >> 4) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((int)threadIdx.x) & 15)) + 229392)] = conv2d_nhwc_local[37];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 112) * 401408) + ((((int)threadIdx.x) >> 5) * 100352)) + (((((int)blockIdx.x) % 112) >> 2) * 512)) + (((((int)threadIdx.x) & 31) >> 4) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((int)threadIdx.x) & 15)) + 229408)] = conv2d_nhwc_local[44];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 112) * 401408) + ((((int)threadIdx.x) >> 5) * 100352)) + (((((int)blockIdx.x) % 112) >> 2) * 512)) + (((((int)threadIdx.x) & 31) >> 4) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((int)threadIdx.x) & 15)) + 229424)] = conv2d_nhwc_local[51];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 112) * 401408) + ((((int)threadIdx.x) >> 5) * 100352)) + (((((int)blockIdx.x) % 112) >> 2) * 512)) + (((((int)threadIdx.x) & 31) >> 4) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((int)threadIdx.x) & 15)) + 43008)] = conv2d_nhwc_local[3];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 112) * 401408) + ((((int)threadIdx.x) >> 5) * 100352)) + (((((int)blockIdx.x) % 112) >> 2) * 512)) + (((((int)threadIdx.x) & 31) >> 4) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((int)threadIdx.x) & 15)) + 43024)] = conv2d_nhwc_local[10];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 112) * 401408) + ((((int)threadIdx.x) >> 5) * 100352)) + (((((int)blockIdx.x) % 112) >> 2) * 512)) + (((((int)threadIdx.x) & 31) >> 4) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((int)threadIdx.x) & 15)) + 43040)] = conv2d_nhwc_local[17];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 112) * 401408) + ((((int)threadIdx.x) >> 5) * 100352)) + (((((int)blockIdx.x) % 112) >> 2) * 512)) + (((((int)threadIdx.x) & 31) >> 4) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((int)threadIdx.x) & 15)) + 43056)] = conv2d_nhwc_local[24];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 112) * 401408) + ((((int)threadIdx.x) >> 5) * 100352)) + (((((int)blockIdx.x) % 112) >> 2) * 512)) + (((((int)threadIdx.x) & 31) >> 4) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((int)threadIdx.x) & 15)) + 243712)] = conv2d_nhwc_local[31];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 112) * 401408) + ((((int)threadIdx.x) >> 5) * 100352)) + (((((int)blockIdx.x) % 112) >> 2) * 512)) + (((((int)threadIdx.x) & 31) >> 4) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((int)threadIdx.x) & 15)) + 243728)] = conv2d_nhwc_local[38];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 112) * 401408) + ((((int)threadIdx.x) >> 5) * 100352)) + (((((int)blockIdx.x) % 112) >> 2) * 512)) + (((((int)threadIdx.x) & 31) >> 4) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((int)threadIdx.x) & 15)) + 243744)] = conv2d_nhwc_local[45];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 112) * 401408) + ((((int)threadIdx.x) >> 5) * 100352)) + (((((int)blockIdx.x) % 112) >> 2) * 512)) + (((((int)threadIdx.x) & 31) >> 4) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((int)threadIdx.x) & 15)) + 243760)] = conv2d_nhwc_local[52];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 112) * 401408) + ((((int)threadIdx.x) >> 5) * 100352)) + (((((int)blockIdx.x) % 112) >> 2) * 512)) + (((((int)threadIdx.x) & 31) >> 4) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((int)threadIdx.x) & 15)) + 57344)] = conv2d_nhwc_local[4];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 112) * 401408) + ((((int)threadIdx.x) >> 5) * 100352)) + (((((int)blockIdx.x) % 112) >> 2) * 512)) + (((((int)threadIdx.x) & 31) >> 4) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((int)threadIdx.x) & 15)) + 57360)] = conv2d_nhwc_local[11];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 112) * 401408) + ((((int)threadIdx.x) >> 5) * 100352)) + (((((int)blockIdx.x) % 112) >> 2) * 512)) + (((((int)threadIdx.x) & 31) >> 4) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((int)threadIdx.x) & 15)) + 57376)] = conv2d_nhwc_local[18];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 112) * 401408) + ((((int)threadIdx.x) >> 5) * 100352)) + (((((int)blockIdx.x) % 112) >> 2) * 512)) + (((((int)threadIdx.x) & 31) >> 4) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((int)threadIdx.x) & 15)) + 57392)] = conv2d_nhwc_local[25];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 112) * 401408) + ((((int)threadIdx.x) >> 5) * 100352)) + (((((int)blockIdx.x) % 112) >> 2) * 512)) + (((((int)threadIdx.x) & 31) >> 4) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((int)threadIdx.x) & 15)) + 258048)] = conv2d_nhwc_local[32];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 112) * 401408) + ((((int)threadIdx.x) >> 5) * 100352)) + (((((int)blockIdx.x) % 112) >> 2) * 512)) + (((((int)threadIdx.x) & 31) >> 4) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((int)threadIdx.x) & 15)) + 258064)] = conv2d_nhwc_local[39];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 112) * 401408) + ((((int)threadIdx.x) >> 5) * 100352)) + (((((int)blockIdx.x) % 112) >> 2) * 512)) + (((((int)threadIdx.x) & 31) >> 4) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((int)threadIdx.x) & 15)) + 258080)] = conv2d_nhwc_local[46];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 112) * 401408) + ((((int)threadIdx.x) >> 5) * 100352)) + (((((int)blockIdx.x) % 112) >> 2) * 512)) + (((((int)threadIdx.x) & 31) >> 4) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((int)threadIdx.x) & 15)) + 258096)] = conv2d_nhwc_local[53];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 112) * 401408) + ((((int)threadIdx.x) >> 5) * 100352)) + (((((int)blockIdx.x) % 112) >> 2) * 512)) + (((((int)threadIdx.x) & 31) >> 4) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((int)threadIdx.x) & 15)) + 71680)] = conv2d_nhwc_local[5];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 112) * 401408) + ((((int)threadIdx.x) >> 5) * 100352)) + (((((int)blockIdx.x) % 112) >> 2) * 512)) + (((((int)threadIdx.x) & 31) >> 4) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((int)threadIdx.x) & 15)) + 71696)] = conv2d_nhwc_local[12];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 112) * 401408) + ((((int)threadIdx.x) >> 5) * 100352)) + (((((int)blockIdx.x) % 112) >> 2) * 512)) + (((((int)threadIdx.x) & 31) >> 4) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((int)threadIdx.x) & 15)) + 71712)] = conv2d_nhwc_local[19];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 112) * 401408) + ((((int)threadIdx.x) >> 5) * 100352)) + (((((int)blockIdx.x) % 112) >> 2) * 512)) + (((((int)threadIdx.x) & 31) >> 4) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((int)threadIdx.x) & 15)) + 71728)] = conv2d_nhwc_local[26];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 112) * 401408) + ((((int)threadIdx.x) >> 5) * 100352)) + (((((int)blockIdx.x) % 112) >> 2) * 512)) + (((((int)threadIdx.x) & 31) >> 4) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((int)threadIdx.x) & 15)) + 272384)] = conv2d_nhwc_local[33];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 112) * 401408) + ((((int)threadIdx.x) >> 5) * 100352)) + (((((int)blockIdx.x) % 112) >> 2) * 512)) + (((((int)threadIdx.x) & 31) >> 4) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((int)threadIdx.x) & 15)) + 272400)] = conv2d_nhwc_local[40];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 112) * 401408) + ((((int)threadIdx.x) >> 5) * 100352)) + (((((int)blockIdx.x) % 112) >> 2) * 512)) + (((((int)threadIdx.x) & 31) >> 4) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((int)threadIdx.x) & 15)) + 272416)] = conv2d_nhwc_local[47];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 112) * 401408) + ((((int)threadIdx.x) >> 5) * 100352)) + (((((int)blockIdx.x) % 112) >> 2) * 512)) + (((((int)threadIdx.x) & 31) >> 4) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((int)threadIdx.x) & 15)) + 272432)] = conv2d_nhwc_local[54];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 112) * 401408) + ((((int)threadIdx.x) >> 5) * 100352)) + (((((int)blockIdx.x) % 112) >> 2) * 512)) + (((((int)threadIdx.x) & 31) >> 4) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((int)threadIdx.x) & 15)) + 86016)] = conv2d_nhwc_local[6];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 112) * 401408) + ((((int)threadIdx.x) >> 5) * 100352)) + (((((int)blockIdx.x) % 112) >> 2) * 512)) + (((((int)threadIdx.x) & 31) >> 4) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((int)threadIdx.x) & 15)) + 86032)] = conv2d_nhwc_local[13];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 112) * 401408) + ((((int)threadIdx.x) >> 5) * 100352)) + (((((int)blockIdx.x) % 112) >> 2) * 512)) + (((((int)threadIdx.x) & 31) >> 4) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((int)threadIdx.x) & 15)) + 86048)] = conv2d_nhwc_local[20];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 112) * 401408) + ((((int)threadIdx.x) >> 5) * 100352)) + (((((int)blockIdx.x) % 112) >> 2) * 512)) + (((((int)threadIdx.x) & 31) >> 4) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((int)threadIdx.x) & 15)) + 86064)] = conv2d_nhwc_local[27];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 112) * 401408) + ((((int)threadIdx.x) >> 5) * 100352)) + (((((int)blockIdx.x) % 112) >> 2) * 512)) + (((((int)threadIdx.x) & 31) >> 4) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((int)threadIdx.x) & 15)) + 286720)] = conv2d_nhwc_local[34];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 112) * 401408) + ((((int)threadIdx.x) >> 5) * 100352)) + (((((int)blockIdx.x) % 112) >> 2) * 512)) + (((((int)threadIdx.x) & 31) >> 4) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((int)threadIdx.x) & 15)) + 286736)] = conv2d_nhwc_local[41];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 112) * 401408) + ((((int)threadIdx.x) >> 5) * 100352)) + (((((int)blockIdx.x) % 112) >> 2) * 512)) + (((((int)threadIdx.x) & 31) >> 4) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((int)threadIdx.x) & 15)) + 286752)] = conv2d_nhwc_local[48];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 112) * 401408) + ((((int)threadIdx.x) >> 5) * 100352)) + (((((int)blockIdx.x) % 112) >> 2) * 512)) + (((((int)threadIdx.x) & 31) >> 4) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((int)threadIdx.x) & 15)) + 286768)] = conv2d_nhwc_local[55];
}


