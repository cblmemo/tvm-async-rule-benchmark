
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(32) main_kernel0(float* __restrict__ inputs, float* __restrict__ weight, float* __restrict__ conv2d_nhwc) {
  float conv2d_nhwc_local[32];
  __shared__ float PadInput_shared[2048];
  __shared__ float weight_shared[2048];
  conv2d_nhwc_local[0] = 0.000000e+00f;
  conv2d_nhwc_local[1] = 0.000000e+00f;
  conv2d_nhwc_local[2] = 0.000000e+00f;
  conv2d_nhwc_local[3] = 0.000000e+00f;
  conv2d_nhwc_local[4] = 0.000000e+00f;
  conv2d_nhwc_local[5] = 0.000000e+00f;
  conv2d_nhwc_local[6] = 0.000000e+00f;
  conv2d_nhwc_local[7] = 0.000000e+00f;
  conv2d_nhwc_local[8] = 0.000000e+00f;
  conv2d_nhwc_local[9] = 0.000000e+00f;
  conv2d_nhwc_local[10] = 0.000000e+00f;
  conv2d_nhwc_local[11] = 0.000000e+00f;
  conv2d_nhwc_local[12] = 0.000000e+00f;
  conv2d_nhwc_local[13] = 0.000000e+00f;
  conv2d_nhwc_local[14] = 0.000000e+00f;
  conv2d_nhwc_local[15] = 0.000000e+00f;
  conv2d_nhwc_local[16] = 0.000000e+00f;
  conv2d_nhwc_local[17] = 0.000000e+00f;
  conv2d_nhwc_local[18] = 0.000000e+00f;
  conv2d_nhwc_local[19] = 0.000000e+00f;
  conv2d_nhwc_local[20] = 0.000000e+00f;
  conv2d_nhwc_local[21] = 0.000000e+00f;
  conv2d_nhwc_local[22] = 0.000000e+00f;
  conv2d_nhwc_local[23] = 0.000000e+00f;
  conv2d_nhwc_local[24] = 0.000000e+00f;
  conv2d_nhwc_local[25] = 0.000000e+00f;
  conv2d_nhwc_local[26] = 0.000000e+00f;
  conv2d_nhwc_local[27] = 0.000000e+00f;
  conv2d_nhwc_local[28] = 0.000000e+00f;
  conv2d_nhwc_local[29] = 0.000000e+00f;
  conv2d_nhwc_local[30] = 0.000000e+00f;
  conv2d_nhwc_local[31] = 0.000000e+00f;

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((int)threadIdx.x) * 4)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((int)blockIdx.x) / 28) * 114688) + ((((int)threadIdx.x) >> 4) * 14336)) + (((((int)blockIdx.x) % 28) >> 1) * 1024)) + (((((int)threadIdx.x) & 15) >> 2) * 256)) + ((((int)threadIdx.x) & 3) * 4)))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 4) + 128)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)blockIdx.x) / 28) * 114688) + ((((int)threadIdx.x) >> 4) * 14336)) + (((((int)blockIdx.x) % 28) >> 1) * 1024)) + (((((int)threadIdx.x) & 15) >> 2) * 256)) + ((((int)threadIdx.x) & 3) * 4)) + 28672))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 4) + 256)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)blockIdx.x) / 28) * 114688) + ((((int)threadIdx.x) >> 4) * 14336)) + (((((int)blockIdx.x) % 28) >> 1) * 1024)) + (((((int)threadIdx.x) & 15) >> 2) * 256)) + ((((int)threadIdx.x) & 3) * 4)) + 57344))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 4) + 384)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)blockIdx.x) / 28) * 114688) + ((((int)threadIdx.x) >> 4) * 14336)) + (((((int)blockIdx.x) % 28) >> 1) * 1024)) + (((((int)threadIdx.x) & 15) >> 2) * 256)) + ((((int)threadIdx.x) & 3) * 4)) + 86016))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) * 4)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((((int)threadIdx.x) >> 3) * 64) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 7) * 4)))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 128)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 64) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 256))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 256)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 64) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 512))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 384)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 64) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 768))), "n"(16)
    );
  }
__asm__ __volatile__("cp.async.commit_group;");


  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 4) + 512)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)blockIdx.x) / 28) * 114688) + ((((int)threadIdx.x) >> 4) * 14336)) + (((((int)blockIdx.x) % 28) >> 1) * 1024)) + (((((int)threadIdx.x) & 15) >> 2) * 256)) + ((((int)threadIdx.x) & 3) * 4)) + 16))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 4) + 640)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)blockIdx.x) / 28) * 114688) + ((((int)threadIdx.x) >> 4) * 14336)) + (((((int)blockIdx.x) % 28) >> 1) * 1024)) + (((((int)threadIdx.x) & 15) >> 2) * 256)) + ((((int)threadIdx.x) & 3) * 4)) + 28688))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 4) + 768)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)blockIdx.x) / 28) * 114688) + ((((int)threadIdx.x) >> 4) * 14336)) + (((((int)blockIdx.x) % 28) >> 1) * 1024)) + (((((int)threadIdx.x) & 15) >> 2) * 256)) + ((((int)threadIdx.x) & 3) * 4)) + 57360))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 4) + 896)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)blockIdx.x) / 28) * 114688) + ((((int)threadIdx.x) >> 4) * 14336)) + (((((int)blockIdx.x) % 28) >> 1) * 1024)) + (((((int)threadIdx.x) & 15) >> 2) * 256)) + ((((int)threadIdx.x) & 3) * 4)) + 86032))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 512)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 64) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 1024))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 640)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 64) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 1280))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 768)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 64) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 1536))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 896)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 64) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 1792))), "n"(16)
    );
  }
__asm__ __volatile__("cp.async.commit_group;");


  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 4) + 1024)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)blockIdx.x) / 28) * 114688) + ((((int)threadIdx.x) >> 4) * 14336)) + (((((int)blockIdx.x) % 28) >> 1) * 1024)) + (((((int)threadIdx.x) & 15) >> 2) * 256)) + ((((int)threadIdx.x) & 3) * 4)) + 32))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 4) + 1152)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)blockIdx.x) / 28) * 114688) + ((((int)threadIdx.x) >> 4) * 14336)) + (((((int)blockIdx.x) % 28) >> 1) * 1024)) + (((((int)threadIdx.x) & 15) >> 2) * 256)) + ((((int)threadIdx.x) & 3) * 4)) + 28704))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 4) + 1280)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)blockIdx.x) / 28) * 114688) + ((((int)threadIdx.x) >> 4) * 14336)) + (((((int)blockIdx.x) % 28) >> 1) * 1024)) + (((((int)threadIdx.x) & 15) >> 2) * 256)) + ((((int)threadIdx.x) & 3) * 4)) + 57376))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 4) + 1408)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)blockIdx.x) / 28) * 114688) + ((((int)threadIdx.x) >> 4) * 14336)) + (((((int)blockIdx.x) % 28) >> 1) * 1024)) + (((((int)threadIdx.x) & 15) >> 2) * 256)) + ((((int)threadIdx.x) & 3) * 4)) + 86048))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 1024)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 64) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 2048))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 1152)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 64) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 2304))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 1280)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 64) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 2560))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 1408)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 64) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 2816))), "n"(16)
    );
  }
__asm__ __volatile__("cp.async.commit_group;");

  for (int rh_0_rw_0_rc_0_fused = 0; rh_0_rw_0_rc_0_fused < 13; ++rh_0_rw_0_rc_0_fused) {
    __syncthreads();

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((rh_0_rw_0_rc_0_fused + 3) & 3) * 512) + (((int)threadIdx.x) * 4))))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((((int)blockIdx.x) / 28) * 114688) + ((((int)threadIdx.x) >> 4) * 14336)) + (((((int)blockIdx.x) % 28) >> 1) * 1024)) + (((((int)threadIdx.x) & 15) >> 2) * 256)) + (rh_0_rw_0_rc_0_fused * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 48))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 512) + (((int)threadIdx.x) * 4)) + 128)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((((int)blockIdx.x) / 28) * 114688) + ((((int)threadIdx.x) >> 4) * 14336)) + (((((int)blockIdx.x) % 28) >> 1) * 1024)) + (((((int)threadIdx.x) & 15) >> 2) * 256)) + (rh_0_rw_0_rc_0_fused * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 28720))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 512) + (((int)threadIdx.x) * 4)) + 256)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((((int)blockIdx.x) / 28) * 114688) + ((((int)threadIdx.x) >> 4) * 14336)) + (((((int)blockIdx.x) % 28) >> 1) * 1024)) + (((((int)threadIdx.x) & 15) >> 2) * 256)) + (rh_0_rw_0_rc_0_fused * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 57392))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 512) + (((int)threadIdx.x) * 4)) + 384)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((((int)blockIdx.x) / 28) * 114688) + ((((int)threadIdx.x) >> 4) * 14336)) + (((((int)blockIdx.x) % 28) >> 1) * 1024)) + (((((int)threadIdx.x) & 15) >> 2) * 256)) + (rh_0_rw_0_rc_0_fused * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 86064))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((rh_0_rw_0_rc_0_fused + 3) & 3) * 512) + (((int)threadIdx.x) * 4))))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 1024) + ((((int)threadIdx.x) >> 3) * 64)) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 3072))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 512) + (((int)threadIdx.x) * 4)) + 128)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 1024) + ((((int)threadIdx.x) >> 3) * 64)) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 3328))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 512) + (((int)threadIdx.x) * 4)) + 256)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 1024) + ((((int)threadIdx.x) >> 3) * 64)) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 3584))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 512) + (((int)threadIdx.x) * 4)) + 384)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 1024) + ((((int)threadIdx.x) >> 3) * 64)) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 3840))), "n"(16)
    );
  }
__asm__ __volatile__("cp.async.commit_group;");

__asm__ __volatile__("cp.async.wait_group 3;");

    __syncthreads();
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16))] * weight_shared[(((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4))]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 1)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 32)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 2)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 64)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 3)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 96)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16))] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 1)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 1)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 33)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 2)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 65)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 3)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 97)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16))] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 2)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 1)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 34)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 2)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 66)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 3)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 98)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16))] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 3)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 1)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 35)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 2)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 67)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 3)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 99)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 64)] * weight_shared[(((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4))]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 65)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 32)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 66)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 64)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 67)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 96)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 64)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 1)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 65)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 33)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 66)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 65)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 67)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 97)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 64)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 2)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 65)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 34)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 66)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 66)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 67)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 98)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 64)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 3)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 65)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 35)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 66)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 67)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 67)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 99)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 128)] * weight_shared[(((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4))]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 129)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 32)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 130)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 64)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 131)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 96)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 128)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 1)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 129)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 33)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 130)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 65)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 131)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 97)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 128)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 2)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 129)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 34)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 130)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 66)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 131)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 98)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 128)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 3)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 129)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 35)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 130)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 67)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 131)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 99)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 192)] * weight_shared[(((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4))]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 193)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 32)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 194)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 64)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 195)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 96)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 192)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 1)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 193)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 33)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 194)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 65)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 195)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 97)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 192)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 2)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 193)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 34)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 194)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 66)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 195)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 98)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 192)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 3)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 193)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 35)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 194)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 67)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 195)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 99)]));
    conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 256)] * weight_shared[(((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4))]));
    conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 257)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 32)]));
    conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 258)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 64)]));
    conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 259)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 96)]));
    conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 256)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 1)]));
    conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 257)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 33)]));
    conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 258)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 65)]));
    conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 259)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 97)]));
    conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 256)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 2)]));
    conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 257)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 34)]));
    conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 258)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 66)]));
    conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 259)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 98)]));
    conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 256)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 3)]));
    conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 257)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 35)]));
    conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 258)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 67)]));
    conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 259)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 99)]));
    conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 320)] * weight_shared[(((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4))]));
    conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 321)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 32)]));
    conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 322)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 64)]));
    conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 323)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 96)]));
    conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 320)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 1)]));
    conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 321)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 33)]));
    conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 322)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 65)]));
    conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 323)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 97)]));
    conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 320)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 2)]));
    conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 321)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 34)]));
    conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 322)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 66)]));
    conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 323)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 98)]));
    conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 320)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 3)]));
    conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 321)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 35)]));
    conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 322)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 67)]));
    conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 323)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 99)]));
    conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 384)] * weight_shared[(((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4))]));
    conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 385)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 32)]));
    conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 386)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 64)]));
    conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 387)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 96)]));
    conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 384)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 1)]));
    conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 385)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 33)]));
    conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 386)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 65)]));
    conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 387)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 97)]));
    conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 384)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 2)]));
    conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 385)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 34)]));
    conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 386)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 66)]));
    conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 387)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 98)]));
    conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 384)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 3)]));
    conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 385)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 35)]));
    conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 386)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 67)]));
    conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 387)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 99)]));
    conv2d_nhwc_local[28] = (conv2d_nhwc_local[28] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 448)] * weight_shared[(((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4))]));
    conv2d_nhwc_local[28] = (conv2d_nhwc_local[28] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 449)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 32)]));
    conv2d_nhwc_local[28] = (conv2d_nhwc_local[28] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 450)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 64)]));
    conv2d_nhwc_local[28] = (conv2d_nhwc_local[28] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 451)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 96)]));
    conv2d_nhwc_local[29] = (conv2d_nhwc_local[29] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 448)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 1)]));
    conv2d_nhwc_local[29] = (conv2d_nhwc_local[29] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 449)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 33)]));
    conv2d_nhwc_local[29] = (conv2d_nhwc_local[29] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 450)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 65)]));
    conv2d_nhwc_local[29] = (conv2d_nhwc_local[29] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 451)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 97)]));
    conv2d_nhwc_local[30] = (conv2d_nhwc_local[30] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 448)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 2)]));
    conv2d_nhwc_local[30] = (conv2d_nhwc_local[30] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 449)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 34)]));
    conv2d_nhwc_local[30] = (conv2d_nhwc_local[30] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 450)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 66)]));
    conv2d_nhwc_local[30] = (conv2d_nhwc_local[30] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 451)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 98)]));
    conv2d_nhwc_local[31] = (conv2d_nhwc_local[31] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 448)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 3)]));
    conv2d_nhwc_local[31] = (conv2d_nhwc_local[31] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 449)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 35)]));
    conv2d_nhwc_local[31] = (conv2d_nhwc_local[31] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 450)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 67)]));
    conv2d_nhwc_local[31] = (conv2d_nhwc_local[31] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 451)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 99)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 4)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 128)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 5)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 160)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 6)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 192)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 7)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 224)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 4)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 129)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 5)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 161)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 6)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 193)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 7)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 225)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 4)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 130)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 5)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 162)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 6)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 194)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 7)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 226)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 4)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 131)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 5)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 163)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 6)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 195)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 7)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 227)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 68)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 128)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 69)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 160)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 70)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 192)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 71)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 224)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 68)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 129)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 69)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 161)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 70)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 193)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 71)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 225)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 68)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 130)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 69)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 162)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 70)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 194)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 71)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 226)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 68)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 131)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 69)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 163)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 70)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 195)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 71)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 227)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 132)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 128)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 133)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 160)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 134)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 192)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 135)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 224)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 132)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 129)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 133)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 161)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 134)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 193)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 135)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 225)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 132)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 130)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 133)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 162)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 134)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 194)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 135)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 226)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 132)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 131)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 133)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 163)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 134)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 195)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 135)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 227)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 196)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 128)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 197)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 160)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 198)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 192)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 199)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 224)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 196)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 129)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 197)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 161)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 198)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 193)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 199)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 225)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 196)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 130)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 197)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 162)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 198)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 194)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 199)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 226)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 196)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 131)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 197)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 163)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 198)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 195)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 199)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 227)]));
    conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 260)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 128)]));
    conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 261)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 160)]));
    conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 262)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 192)]));
    conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 263)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 224)]));
    conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 260)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 129)]));
    conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 261)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 161)]));
    conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 262)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 193)]));
    conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 263)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 225)]));
    conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 260)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 130)]));
    conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 261)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 162)]));
    conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 262)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 194)]));
    conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 263)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 226)]));
    conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 260)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 131)]));
    conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 261)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 163)]));
    conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 262)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 195)]));
    conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 263)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 227)]));
    conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 324)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 128)]));
    conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 325)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 160)]));
    conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 326)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 192)]));
    conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 327)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 224)]));
    conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 324)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 129)]));
    conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 325)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 161)]));
    conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 326)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 193)]));
    conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 327)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 225)]));
    conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 324)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 130)]));
    conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 325)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 162)]));
    conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 326)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 194)]));
    conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 327)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 226)]));
    conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 324)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 131)]));
    conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 325)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 163)]));
    conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 326)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 195)]));
    conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 327)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 227)]));
    conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 388)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 128)]));
    conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 389)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 160)]));
    conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 390)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 192)]));
    conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 391)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 224)]));
    conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 388)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 129)]));
    conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 389)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 161)]));
    conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 390)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 193)]));
    conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 391)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 225)]));
    conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 388)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 130)]));
    conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 389)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 162)]));
    conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 390)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 194)]));
    conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 391)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 226)]));
    conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 388)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 131)]));
    conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 389)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 163)]));
    conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 390)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 195)]));
    conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 391)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 227)]));
    conv2d_nhwc_local[28] = (conv2d_nhwc_local[28] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 452)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 128)]));
    conv2d_nhwc_local[28] = (conv2d_nhwc_local[28] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 453)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 160)]));
    conv2d_nhwc_local[28] = (conv2d_nhwc_local[28] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 454)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 192)]));
    conv2d_nhwc_local[28] = (conv2d_nhwc_local[28] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 455)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 224)]));
    conv2d_nhwc_local[29] = (conv2d_nhwc_local[29] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 452)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 129)]));
    conv2d_nhwc_local[29] = (conv2d_nhwc_local[29] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 453)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 161)]));
    conv2d_nhwc_local[29] = (conv2d_nhwc_local[29] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 454)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 193)]));
    conv2d_nhwc_local[29] = (conv2d_nhwc_local[29] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 455)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 225)]));
    conv2d_nhwc_local[30] = (conv2d_nhwc_local[30] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 452)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 130)]));
    conv2d_nhwc_local[30] = (conv2d_nhwc_local[30] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 453)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 162)]));
    conv2d_nhwc_local[30] = (conv2d_nhwc_local[30] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 454)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 194)]));
    conv2d_nhwc_local[30] = (conv2d_nhwc_local[30] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 455)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 226)]));
    conv2d_nhwc_local[31] = (conv2d_nhwc_local[31] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 452)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 131)]));
    conv2d_nhwc_local[31] = (conv2d_nhwc_local[31] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 453)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 163)]));
    conv2d_nhwc_local[31] = (conv2d_nhwc_local[31] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 454)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 195)]));
    conv2d_nhwc_local[31] = (conv2d_nhwc_local[31] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 455)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 227)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 8)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 256)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 9)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 288)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 10)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 320)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 11)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 352)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 8)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 257)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 9)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 289)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 10)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 321)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 11)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 353)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 8)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 258)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 9)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 290)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 10)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 322)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 11)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 354)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 8)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 259)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 9)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 291)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 10)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 323)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 11)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 355)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 72)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 256)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 73)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 288)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 74)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 320)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 75)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 352)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 72)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 257)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 73)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 289)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 74)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 321)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 75)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 353)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 72)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 258)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 73)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 290)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 74)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 322)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 75)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 354)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 72)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 259)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 73)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 291)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 74)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 323)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 75)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 355)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 136)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 256)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 137)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 288)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 138)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 320)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 139)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 352)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 136)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 257)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 137)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 289)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 138)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 321)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 139)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 353)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 136)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 258)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 137)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 290)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 138)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 322)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 139)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 354)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 136)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 259)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 137)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 291)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 138)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 323)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 139)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 355)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 200)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 256)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 201)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 288)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 202)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 320)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 203)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 352)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 200)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 257)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 201)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 289)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 202)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 321)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 203)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 353)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 200)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 258)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 201)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 290)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 202)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 322)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 203)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 354)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 200)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 259)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 201)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 291)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 202)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 323)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 203)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 355)]));
    conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 264)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 256)]));
    conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 265)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 288)]));
    conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 266)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 320)]));
    conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 267)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 352)]));
    conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 264)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 257)]));
    conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 265)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 289)]));
    conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 266)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 321)]));
    conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 267)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 353)]));
    conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 264)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 258)]));
    conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 265)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 290)]));
    conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 266)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 322)]));
    conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 267)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 354)]));
    conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 264)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 259)]));
    conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 265)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 291)]));
    conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 266)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 323)]));
    conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 267)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 355)]));
    conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 328)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 256)]));
    conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 329)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 288)]));
    conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 330)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 320)]));
    conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 331)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 352)]));
    conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 328)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 257)]));
    conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 329)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 289)]));
    conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 330)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 321)]));
    conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 331)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 353)]));
    conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 328)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 258)]));
    conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 329)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 290)]));
    conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 330)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 322)]));
    conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 331)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 354)]));
    conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 328)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 259)]));
    conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 329)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 291)]));
    conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 330)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 323)]));
    conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 331)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 355)]));
    conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 392)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 256)]));
    conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 393)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 288)]));
    conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 394)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 320)]));
    conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 395)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 352)]));
    conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 392)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 257)]));
    conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 393)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 289)]));
    conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 394)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 321)]));
    conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 395)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 353)]));
    conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 392)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 258)]));
    conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 393)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 290)]));
    conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 394)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 322)]));
    conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 395)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 354)]));
    conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 392)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 259)]));
    conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 393)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 291)]));
    conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 394)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 323)]));
    conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 395)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 355)]));
    conv2d_nhwc_local[28] = (conv2d_nhwc_local[28] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 456)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 256)]));
    conv2d_nhwc_local[28] = (conv2d_nhwc_local[28] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 457)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 288)]));
    conv2d_nhwc_local[28] = (conv2d_nhwc_local[28] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 458)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 320)]));
    conv2d_nhwc_local[28] = (conv2d_nhwc_local[28] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 459)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 352)]));
    conv2d_nhwc_local[29] = (conv2d_nhwc_local[29] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 456)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 257)]));
    conv2d_nhwc_local[29] = (conv2d_nhwc_local[29] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 457)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 289)]));
    conv2d_nhwc_local[29] = (conv2d_nhwc_local[29] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 458)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 321)]));
    conv2d_nhwc_local[29] = (conv2d_nhwc_local[29] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 459)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 353)]));
    conv2d_nhwc_local[30] = (conv2d_nhwc_local[30] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 456)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 258)]));
    conv2d_nhwc_local[30] = (conv2d_nhwc_local[30] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 457)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 290)]));
    conv2d_nhwc_local[30] = (conv2d_nhwc_local[30] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 458)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 322)]));
    conv2d_nhwc_local[30] = (conv2d_nhwc_local[30] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 459)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 354)]));
    conv2d_nhwc_local[31] = (conv2d_nhwc_local[31] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 456)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 259)]));
    conv2d_nhwc_local[31] = (conv2d_nhwc_local[31] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 457)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 291)]));
    conv2d_nhwc_local[31] = (conv2d_nhwc_local[31] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 458)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 323)]));
    conv2d_nhwc_local[31] = (conv2d_nhwc_local[31] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 459)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 355)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 12)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 384)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 13)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 416)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 14)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 448)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 15)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 480)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 12)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 385)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 13)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 417)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 14)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 449)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 15)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 481)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 12)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 386)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 13)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 418)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 14)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 450)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 15)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 482)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 12)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 387)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 13)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 419)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 14)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 451)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 15)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 483)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 76)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 384)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 77)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 416)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 78)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 448)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 79)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 480)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 76)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 385)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 77)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 417)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 78)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 449)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 79)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 481)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 76)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 386)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 77)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 418)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 78)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 450)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 79)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 482)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 76)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 387)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 77)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 419)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 78)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 451)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 79)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 483)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 140)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 384)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 141)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 416)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 142)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 448)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 143)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 480)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 140)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 385)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 141)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 417)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 142)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 449)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 143)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 481)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 140)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 386)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 141)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 418)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 142)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 450)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 143)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 482)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 140)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 387)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 141)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 419)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 142)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 451)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 143)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 483)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 204)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 384)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 205)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 416)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 206)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 448)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 207)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 480)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 204)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 385)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 205)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 417)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 206)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 449)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 207)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 481)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 204)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 386)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 205)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 418)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 206)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 450)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 207)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 482)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 204)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 387)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 205)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 419)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 206)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 451)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 207)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 483)]));
    conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 268)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 384)]));
    conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 269)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 416)]));
    conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 270)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 448)]));
    conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 271)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 480)]));
    conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 268)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 385)]));
    conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 269)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 417)]));
    conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 270)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 449)]));
    conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 271)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 481)]));
    conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 268)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 386)]));
    conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 269)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 418)]));
    conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 270)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 450)]));
    conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 271)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 482)]));
    conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 268)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 387)]));
    conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 269)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 419)]));
    conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 270)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 451)]));
    conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 271)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 483)]));
    conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 332)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 384)]));
    conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 333)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 416)]));
    conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 334)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 448)]));
    conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 335)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 480)]));
    conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 332)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 385)]));
    conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 333)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 417)]));
    conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 334)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 449)]));
    conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 335)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 481)]));
    conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 332)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 386)]));
    conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 333)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 418)]));
    conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 334)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 450)]));
    conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 335)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 482)]));
    conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 332)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 387)]));
    conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 333)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 419)]));
    conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 334)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 451)]));
    conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 335)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 483)]));
    conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 396)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 384)]));
    conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 397)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 416)]));
    conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 398)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 448)]));
    conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 399)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 480)]));
    conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 396)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 385)]));
    conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 397)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 417)]));
    conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 398)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 449)]));
    conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 399)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 481)]));
    conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 396)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 386)]));
    conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 397)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 418)]));
    conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 398)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 450)]));
    conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 399)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 482)]));
    conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 396)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 387)]));
    conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 397)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 419)]));
    conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 398)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 451)]));
    conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 399)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 483)]));
    conv2d_nhwc_local[28] = (conv2d_nhwc_local[28] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 460)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 384)]));
    conv2d_nhwc_local[28] = (conv2d_nhwc_local[28] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 461)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 416)]));
    conv2d_nhwc_local[28] = (conv2d_nhwc_local[28] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 462)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 448)]));
    conv2d_nhwc_local[28] = (conv2d_nhwc_local[28] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 463)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 480)]));
    conv2d_nhwc_local[29] = (conv2d_nhwc_local[29] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 460)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 385)]));
    conv2d_nhwc_local[29] = (conv2d_nhwc_local[29] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 461)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 417)]));
    conv2d_nhwc_local[29] = (conv2d_nhwc_local[29] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 462)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 449)]));
    conv2d_nhwc_local[29] = (conv2d_nhwc_local[29] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 463)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 481)]));
    conv2d_nhwc_local[30] = (conv2d_nhwc_local[30] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 460)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 386)]));
    conv2d_nhwc_local[30] = (conv2d_nhwc_local[30] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 461)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 418)]));
    conv2d_nhwc_local[30] = (conv2d_nhwc_local[30] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 462)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 450)]));
    conv2d_nhwc_local[30] = (conv2d_nhwc_local[30] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 463)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 482)]));
    conv2d_nhwc_local[31] = (conv2d_nhwc_local[31] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 460)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 387)]));
    conv2d_nhwc_local[31] = (conv2d_nhwc_local[31] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 461)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 419)]));
    conv2d_nhwc_local[31] = (conv2d_nhwc_local[31] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 462)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 451)]));
    conv2d_nhwc_local[31] = (conv2d_nhwc_local[31] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 3) * 16)) + 463)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 483)]));
  }
__asm__ __volatile__("cp.async.wait_group 2;");

  __syncthreads();
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 512)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 512)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 513)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 544)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 514)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 576)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 515)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 608)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 512)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 513)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 513)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 545)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 514)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 577)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 515)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 609)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 512)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 514)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 513)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 546)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 514)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 578)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 515)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 610)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 512)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 515)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 513)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 547)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 514)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 579)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 515)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 611)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 576)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 512)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 577)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 544)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 578)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 576)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 579)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 608)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 576)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 513)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 577)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 545)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 578)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 577)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 579)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 609)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 576)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 514)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 577)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 546)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 578)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 578)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 579)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 610)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 576)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 515)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 577)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 547)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 578)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 579)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 579)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 611)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 640)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 512)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 641)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 544)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 642)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 576)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 643)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 608)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 640)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 513)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 641)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 545)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 642)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 577)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 643)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 609)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 640)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 514)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 641)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 546)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 642)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 578)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 643)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 610)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 640)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 515)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 641)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 547)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 642)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 579)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 643)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 611)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 704)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 512)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 705)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 544)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 706)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 576)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 707)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 608)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 704)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 513)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 705)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 545)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 706)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 577)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 707)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 609)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 704)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 514)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 705)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 546)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 706)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 578)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 707)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 610)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 704)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 515)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 705)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 547)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 706)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 579)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 707)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 611)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 768)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 512)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 769)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 544)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 770)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 576)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 771)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 608)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 768)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 513)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 769)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 545)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 770)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 577)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 771)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 609)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 768)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 514)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 769)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 546)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 770)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 578)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 771)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 610)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 768)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 515)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 769)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 547)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 770)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 579)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 771)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 611)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 832)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 512)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 833)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 544)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 834)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 576)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 835)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 608)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 832)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 513)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 833)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 545)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 834)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 577)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 835)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 609)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 832)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 514)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 833)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 546)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 834)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 578)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 835)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 610)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 832)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 515)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 833)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 547)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 834)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 579)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 835)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 611)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 896)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 512)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 897)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 544)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 898)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 576)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 899)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 608)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 896)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 513)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 897)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 545)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 898)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 577)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 899)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 609)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 896)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 514)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 897)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 546)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 898)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 578)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 899)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 610)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 896)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 515)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 897)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 547)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 898)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 579)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 899)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 611)]));
  conv2d_nhwc_local[28] = (conv2d_nhwc_local[28] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 960)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 512)]));
  conv2d_nhwc_local[28] = (conv2d_nhwc_local[28] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 961)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 544)]));
  conv2d_nhwc_local[28] = (conv2d_nhwc_local[28] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 962)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 576)]));
  conv2d_nhwc_local[28] = (conv2d_nhwc_local[28] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 963)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 608)]));
  conv2d_nhwc_local[29] = (conv2d_nhwc_local[29] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 960)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 513)]));
  conv2d_nhwc_local[29] = (conv2d_nhwc_local[29] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 961)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 545)]));
  conv2d_nhwc_local[29] = (conv2d_nhwc_local[29] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 962)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 577)]));
  conv2d_nhwc_local[29] = (conv2d_nhwc_local[29] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 963)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 609)]));
  conv2d_nhwc_local[30] = (conv2d_nhwc_local[30] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 960)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 514)]));
  conv2d_nhwc_local[30] = (conv2d_nhwc_local[30] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 961)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 546)]));
  conv2d_nhwc_local[30] = (conv2d_nhwc_local[30] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 962)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 578)]));
  conv2d_nhwc_local[30] = (conv2d_nhwc_local[30] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 963)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 610)]));
  conv2d_nhwc_local[31] = (conv2d_nhwc_local[31] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 960)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 515)]));
  conv2d_nhwc_local[31] = (conv2d_nhwc_local[31] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 961)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 547)]));
  conv2d_nhwc_local[31] = (conv2d_nhwc_local[31] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 962)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 579)]));
  conv2d_nhwc_local[31] = (conv2d_nhwc_local[31] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 963)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 611)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 516)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 640)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 517)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 672)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 518)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 704)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 519)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 736)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 516)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 641)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 517)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 673)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 518)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 705)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 519)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 737)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 516)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 642)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 517)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 674)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 518)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 706)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 519)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 738)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 516)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 643)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 517)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 675)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 518)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 707)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 519)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 739)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 580)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 640)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 581)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 672)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 582)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 704)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 583)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 736)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 580)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 641)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 581)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 673)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 582)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 705)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 583)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 737)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 580)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 642)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 581)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 674)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 582)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 706)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 583)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 738)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 580)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 643)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 581)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 675)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 582)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 707)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 583)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 739)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 644)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 640)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 645)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 672)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 646)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 704)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 647)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 736)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 644)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 641)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 645)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 673)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 646)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 705)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 647)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 737)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 644)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 642)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 645)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 674)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 646)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 706)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 647)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 738)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 644)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 643)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 645)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 675)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 646)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 707)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 647)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 739)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 708)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 640)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 709)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 672)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 710)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 704)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 711)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 736)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 708)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 641)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 709)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 673)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 710)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 705)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 711)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 737)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 708)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 642)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 709)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 674)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 710)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 706)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 711)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 738)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 708)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 643)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 709)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 675)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 710)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 707)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 711)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 739)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 772)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 640)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 773)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 672)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 774)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 704)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 775)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 736)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 772)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 641)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 773)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 673)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 774)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 705)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 775)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 737)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 772)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 642)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 773)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 674)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 774)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 706)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 775)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 738)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 772)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 643)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 773)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 675)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 774)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 707)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 775)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 739)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 836)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 640)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 837)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 672)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 838)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 704)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 839)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 736)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 836)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 641)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 837)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 673)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 838)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 705)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 839)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 737)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 836)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 642)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 837)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 674)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 838)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 706)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 839)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 738)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 836)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 643)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 837)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 675)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 838)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 707)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 839)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 739)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 900)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 640)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 901)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 672)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 902)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 704)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 903)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 736)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 900)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 641)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 901)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 673)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 902)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 705)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 903)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 737)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 900)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 642)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 901)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 674)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 902)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 706)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 903)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 738)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 900)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 643)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 901)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 675)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 902)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 707)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 903)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 739)]));
  conv2d_nhwc_local[28] = (conv2d_nhwc_local[28] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 964)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 640)]));
  conv2d_nhwc_local[28] = (conv2d_nhwc_local[28] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 965)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 672)]));
  conv2d_nhwc_local[28] = (conv2d_nhwc_local[28] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 966)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 704)]));
  conv2d_nhwc_local[28] = (conv2d_nhwc_local[28] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 967)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 736)]));
  conv2d_nhwc_local[29] = (conv2d_nhwc_local[29] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 964)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 641)]));
  conv2d_nhwc_local[29] = (conv2d_nhwc_local[29] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 965)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 673)]));
  conv2d_nhwc_local[29] = (conv2d_nhwc_local[29] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 966)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 705)]));
  conv2d_nhwc_local[29] = (conv2d_nhwc_local[29] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 967)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 737)]));
  conv2d_nhwc_local[30] = (conv2d_nhwc_local[30] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 964)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 642)]));
  conv2d_nhwc_local[30] = (conv2d_nhwc_local[30] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 965)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 674)]));
  conv2d_nhwc_local[30] = (conv2d_nhwc_local[30] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 966)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 706)]));
  conv2d_nhwc_local[30] = (conv2d_nhwc_local[30] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 967)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 738)]));
  conv2d_nhwc_local[31] = (conv2d_nhwc_local[31] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 964)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 643)]));
  conv2d_nhwc_local[31] = (conv2d_nhwc_local[31] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 965)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 675)]));
  conv2d_nhwc_local[31] = (conv2d_nhwc_local[31] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 966)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 707)]));
  conv2d_nhwc_local[31] = (conv2d_nhwc_local[31] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 967)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 739)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 520)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 768)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 521)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 800)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 522)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 832)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 523)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 864)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 520)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 769)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 521)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 801)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 522)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 833)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 523)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 865)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 520)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 770)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 521)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 802)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 522)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 834)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 523)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 866)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 520)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 771)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 521)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 803)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 522)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 835)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 523)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 867)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 584)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 768)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 585)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 800)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 586)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 832)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 587)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 864)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 584)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 769)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 585)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 801)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 586)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 833)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 587)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 865)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 584)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 770)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 585)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 802)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 586)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 834)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 587)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 866)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 584)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 771)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 585)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 803)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 586)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 835)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 587)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 867)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 648)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 768)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 649)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 800)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 650)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 832)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 651)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 864)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 648)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 769)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 649)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 801)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 650)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 833)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 651)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 865)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 648)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 770)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 649)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 802)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 650)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 834)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 651)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 866)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 648)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 771)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 649)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 803)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 650)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 835)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 651)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 867)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 712)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 768)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 713)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 800)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 714)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 832)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 715)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 864)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 712)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 769)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 713)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 801)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 714)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 833)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 715)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 865)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 712)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 770)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 713)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 802)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 714)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 834)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 715)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 866)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 712)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 771)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 713)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 803)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 714)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 835)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 715)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 867)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 776)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 768)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 777)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 800)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 778)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 832)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 779)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 864)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 776)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 769)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 777)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 801)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 778)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 833)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 779)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 865)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 776)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 770)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 777)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 802)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 778)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 834)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 779)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 866)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 776)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 771)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 777)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 803)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 778)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 835)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 779)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 867)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 840)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 768)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 841)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 800)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 842)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 832)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 843)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 864)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 840)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 769)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 841)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 801)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 842)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 833)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 843)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 865)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 840)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 770)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 841)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 802)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 842)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 834)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 843)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 866)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 840)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 771)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 841)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 803)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 842)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 835)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 843)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 867)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 904)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 768)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 905)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 800)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 906)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 832)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 907)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 864)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 904)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 769)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 905)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 801)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 906)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 833)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 907)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 865)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 904)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 770)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 905)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 802)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 906)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 834)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 907)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 866)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 904)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 771)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 905)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 803)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 906)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 835)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 907)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 867)]));
  conv2d_nhwc_local[28] = (conv2d_nhwc_local[28] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 968)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 768)]));
  conv2d_nhwc_local[28] = (conv2d_nhwc_local[28] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 969)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 800)]));
  conv2d_nhwc_local[28] = (conv2d_nhwc_local[28] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 970)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 832)]));
  conv2d_nhwc_local[28] = (conv2d_nhwc_local[28] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 971)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 864)]));
  conv2d_nhwc_local[29] = (conv2d_nhwc_local[29] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 968)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 769)]));
  conv2d_nhwc_local[29] = (conv2d_nhwc_local[29] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 969)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 801)]));
  conv2d_nhwc_local[29] = (conv2d_nhwc_local[29] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 970)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 833)]));
  conv2d_nhwc_local[29] = (conv2d_nhwc_local[29] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 971)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 865)]));
  conv2d_nhwc_local[30] = (conv2d_nhwc_local[30] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 968)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 770)]));
  conv2d_nhwc_local[30] = (conv2d_nhwc_local[30] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 969)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 802)]));
  conv2d_nhwc_local[30] = (conv2d_nhwc_local[30] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 970)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 834)]));
  conv2d_nhwc_local[30] = (conv2d_nhwc_local[30] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 971)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 866)]));
  conv2d_nhwc_local[31] = (conv2d_nhwc_local[31] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 968)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 771)]));
  conv2d_nhwc_local[31] = (conv2d_nhwc_local[31] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 969)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 803)]));
  conv2d_nhwc_local[31] = (conv2d_nhwc_local[31] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 970)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 835)]));
  conv2d_nhwc_local[31] = (conv2d_nhwc_local[31] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 971)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 867)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 524)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 896)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 525)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 928)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 526)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 960)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 527)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 992)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 524)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 897)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 525)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 929)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 526)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 961)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 527)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 993)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 524)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 898)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 525)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 930)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 526)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 962)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 527)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 994)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 524)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 899)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 525)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 931)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 526)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 963)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 527)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 995)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 588)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 896)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 589)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 928)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 590)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 960)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 591)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 992)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 588)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 897)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 589)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 929)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 590)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 961)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 591)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 993)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 588)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 898)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 589)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 930)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 590)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 962)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 591)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 994)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 588)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 899)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 589)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 931)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 590)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 963)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 591)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 995)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 652)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 896)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 653)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 928)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 654)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 960)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 655)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 992)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 652)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 897)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 653)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 929)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 654)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 961)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 655)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 993)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 652)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 898)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 653)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 930)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 654)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 962)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 655)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 994)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 652)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 899)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 653)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 931)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 654)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 963)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 655)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 995)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 716)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 896)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 717)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 928)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 718)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 960)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 719)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 992)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 716)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 897)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 717)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 929)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 718)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 961)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 719)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 993)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 716)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 898)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 717)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 930)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 718)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 962)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 719)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 994)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 716)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 899)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 717)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 931)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 718)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 963)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 719)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 995)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 780)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 896)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 781)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 928)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 782)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 960)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 783)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 992)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 780)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 897)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 781)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 929)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 782)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 961)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 783)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 993)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 780)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 898)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 781)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 930)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 782)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 962)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 783)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 994)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 780)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 899)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 781)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 931)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 782)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 963)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 783)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 995)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 844)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 896)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 845)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 928)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 846)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 960)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 847)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 992)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 844)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 897)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 845)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 929)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 846)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 961)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 847)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 993)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 844)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 898)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 845)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 930)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 846)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 962)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 847)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 994)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 844)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 899)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 845)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 931)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 846)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 963)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 847)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 995)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 908)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 896)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 909)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 928)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 910)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 960)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 911)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 992)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 908)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 897)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 909)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 929)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 910)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 961)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 911)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 993)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 908)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 898)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 909)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 930)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 910)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 962)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 911)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 994)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 908)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 899)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 909)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 931)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 910)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 963)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 911)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 995)]));
  conv2d_nhwc_local[28] = (conv2d_nhwc_local[28] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 972)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 896)]));
  conv2d_nhwc_local[28] = (conv2d_nhwc_local[28] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 973)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 928)]));
  conv2d_nhwc_local[28] = (conv2d_nhwc_local[28] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 974)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 960)]));
  conv2d_nhwc_local[28] = (conv2d_nhwc_local[28] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 975)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 992)]));
  conv2d_nhwc_local[29] = (conv2d_nhwc_local[29] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 972)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 897)]));
  conv2d_nhwc_local[29] = (conv2d_nhwc_local[29] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 973)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 929)]));
  conv2d_nhwc_local[29] = (conv2d_nhwc_local[29] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 974)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 961)]));
  conv2d_nhwc_local[29] = (conv2d_nhwc_local[29] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 975)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 993)]));
  conv2d_nhwc_local[30] = (conv2d_nhwc_local[30] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 972)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 898)]));
  conv2d_nhwc_local[30] = (conv2d_nhwc_local[30] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 973)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 930)]));
  conv2d_nhwc_local[30] = (conv2d_nhwc_local[30] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 974)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 962)]));
  conv2d_nhwc_local[30] = (conv2d_nhwc_local[30] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 975)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 994)]));
  conv2d_nhwc_local[31] = (conv2d_nhwc_local[31] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 972)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 899)]));
  conv2d_nhwc_local[31] = (conv2d_nhwc_local[31] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 973)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 931)]));
  conv2d_nhwc_local[31] = (conv2d_nhwc_local[31] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 974)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 963)]));
  conv2d_nhwc_local[31] = (conv2d_nhwc_local[31] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 975)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 995)]));
__asm__ __volatile__("cp.async.wait_group 1;");

  __syncthreads();
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1024)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1024)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1025)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1056)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1026)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1088)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1027)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1120)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1024)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1025)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1025)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1057)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1026)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1089)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1027)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1121)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1024)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1026)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1025)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1058)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1026)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1090)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1027)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1122)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1024)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1027)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1025)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1059)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1026)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1091)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1027)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1123)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1088)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1024)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1089)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1056)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1090)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1088)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1091)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1120)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1088)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1025)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1089)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1057)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1090)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1089)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1091)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1121)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1088)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1026)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1089)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1058)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1090)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1090)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1091)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1122)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1088)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1027)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1089)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1059)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1090)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1091)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1091)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1123)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1152)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1024)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1153)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1056)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1154)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1088)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1155)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1120)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1152)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1025)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1153)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1057)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1154)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1089)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1155)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1121)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1152)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1026)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1153)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1058)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1154)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1090)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1155)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1122)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1152)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1027)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1153)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1059)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1154)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1091)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1155)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1123)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1216)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1024)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1217)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1056)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1218)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1088)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1219)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1120)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1216)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1025)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1217)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1057)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1218)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1089)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1219)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1121)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1216)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1026)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1217)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1058)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1218)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1090)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1219)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1122)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1216)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1027)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1217)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1059)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1218)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1091)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1219)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1123)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1280)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1024)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1281)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1056)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1282)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1088)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1283)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1120)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1280)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1025)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1281)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1057)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1282)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1089)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1283)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1121)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1280)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1026)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1281)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1058)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1282)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1090)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1283)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1122)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1280)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1027)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1281)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1059)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1282)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1091)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1283)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1123)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1344)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1024)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1345)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1056)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1346)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1088)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1347)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1120)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1344)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1025)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1345)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1057)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1346)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1089)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1347)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1121)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1344)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1026)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1345)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1058)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1346)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1090)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1347)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1122)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1344)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1027)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1345)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1059)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1346)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1091)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1347)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1123)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1408)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1024)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1409)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1056)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1410)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1088)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1411)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1120)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1408)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1025)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1409)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1057)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1410)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1089)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1411)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1121)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1408)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1026)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1409)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1058)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1410)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1090)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1411)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1122)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1408)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1027)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1409)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1059)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1410)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1091)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1411)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1123)]));
  conv2d_nhwc_local[28] = (conv2d_nhwc_local[28] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1472)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1024)]));
  conv2d_nhwc_local[28] = (conv2d_nhwc_local[28] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1473)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1056)]));
  conv2d_nhwc_local[28] = (conv2d_nhwc_local[28] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1474)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1088)]));
  conv2d_nhwc_local[28] = (conv2d_nhwc_local[28] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1475)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1120)]));
  conv2d_nhwc_local[29] = (conv2d_nhwc_local[29] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1472)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1025)]));
  conv2d_nhwc_local[29] = (conv2d_nhwc_local[29] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1473)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1057)]));
  conv2d_nhwc_local[29] = (conv2d_nhwc_local[29] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1474)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1089)]));
  conv2d_nhwc_local[29] = (conv2d_nhwc_local[29] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1475)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1121)]));
  conv2d_nhwc_local[30] = (conv2d_nhwc_local[30] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1472)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1026)]));
  conv2d_nhwc_local[30] = (conv2d_nhwc_local[30] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1473)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1058)]));
  conv2d_nhwc_local[30] = (conv2d_nhwc_local[30] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1474)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1090)]));
  conv2d_nhwc_local[30] = (conv2d_nhwc_local[30] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1475)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1122)]));
  conv2d_nhwc_local[31] = (conv2d_nhwc_local[31] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1472)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1027)]));
  conv2d_nhwc_local[31] = (conv2d_nhwc_local[31] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1473)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1059)]));
  conv2d_nhwc_local[31] = (conv2d_nhwc_local[31] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1474)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1091)]));
  conv2d_nhwc_local[31] = (conv2d_nhwc_local[31] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1475)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1123)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1028)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1152)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1029)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1184)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1030)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1216)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1031)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1248)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1028)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1153)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1029)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1185)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1030)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1217)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1031)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1249)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1028)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1154)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1029)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1186)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1030)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1218)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1031)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1250)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1028)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1155)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1029)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1187)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1030)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1219)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1031)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1251)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1092)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1152)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1093)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1184)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1094)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1216)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1095)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1248)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1092)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1153)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1093)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1185)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1094)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1217)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1095)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1249)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1092)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1154)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1093)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1186)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1094)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1218)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1095)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1250)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1092)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1155)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1093)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1187)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1094)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1219)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1095)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1251)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1156)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1152)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1157)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1184)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1158)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1216)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1159)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1248)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1156)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1153)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1157)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1185)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1158)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1217)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1159)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1249)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1156)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1154)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1157)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1186)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1158)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1218)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1159)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1250)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1156)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1155)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1157)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1187)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1158)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1219)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1159)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1251)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1220)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1152)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1221)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1184)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1222)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1216)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1223)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1248)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1220)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1153)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1221)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1185)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1222)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1217)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1223)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1249)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1220)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1154)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1221)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1186)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1222)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1218)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1223)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1250)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1220)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1155)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1221)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1187)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1222)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1219)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1223)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1251)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1284)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1152)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1285)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1184)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1286)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1216)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1287)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1248)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1284)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1153)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1285)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1185)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1286)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1217)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1287)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1249)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1284)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1154)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1285)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1186)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1286)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1218)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1287)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1250)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1284)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1155)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1285)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1187)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1286)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1219)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1287)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1251)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1348)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1152)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1349)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1184)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1350)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1216)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1351)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1248)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1348)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1153)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1349)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1185)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1350)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1217)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1351)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1249)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1348)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1154)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1349)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1186)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1350)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1218)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1351)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1250)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1348)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1155)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1349)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1187)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1350)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1219)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1351)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1251)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1412)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1152)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1413)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1184)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1414)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1216)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1415)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1248)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1412)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1153)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1413)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1185)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1414)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1217)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1415)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1249)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1412)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1154)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1413)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1186)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1414)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1218)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1415)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1250)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1412)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1155)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1413)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1187)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1414)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1219)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1415)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1251)]));
  conv2d_nhwc_local[28] = (conv2d_nhwc_local[28] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1476)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1152)]));
  conv2d_nhwc_local[28] = (conv2d_nhwc_local[28] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1477)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1184)]));
  conv2d_nhwc_local[28] = (conv2d_nhwc_local[28] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1478)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1216)]));
  conv2d_nhwc_local[28] = (conv2d_nhwc_local[28] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1479)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1248)]));
  conv2d_nhwc_local[29] = (conv2d_nhwc_local[29] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1476)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1153)]));
  conv2d_nhwc_local[29] = (conv2d_nhwc_local[29] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1477)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1185)]));
  conv2d_nhwc_local[29] = (conv2d_nhwc_local[29] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1478)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1217)]));
  conv2d_nhwc_local[29] = (conv2d_nhwc_local[29] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1479)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1249)]));
  conv2d_nhwc_local[30] = (conv2d_nhwc_local[30] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1476)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1154)]));
  conv2d_nhwc_local[30] = (conv2d_nhwc_local[30] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1477)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1186)]));
  conv2d_nhwc_local[30] = (conv2d_nhwc_local[30] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1478)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1218)]));
  conv2d_nhwc_local[30] = (conv2d_nhwc_local[30] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1479)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1250)]));
  conv2d_nhwc_local[31] = (conv2d_nhwc_local[31] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1476)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1155)]));
  conv2d_nhwc_local[31] = (conv2d_nhwc_local[31] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1477)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1187)]));
  conv2d_nhwc_local[31] = (conv2d_nhwc_local[31] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1478)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1219)]));
  conv2d_nhwc_local[31] = (conv2d_nhwc_local[31] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1479)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1251)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1032)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1280)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1033)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1312)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1034)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1344)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1035)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1376)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1032)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1281)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1033)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1313)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1034)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1345)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1035)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1377)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1032)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1282)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1033)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1314)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1034)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1346)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1035)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1378)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1032)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1283)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1033)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1315)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1034)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1347)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1035)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1379)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1096)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1280)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1097)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1312)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1098)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1344)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1099)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1376)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1096)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1281)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1097)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1313)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1098)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1345)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1099)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1377)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1096)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1282)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1097)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1314)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1098)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1346)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1099)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1378)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1096)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1283)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1097)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1315)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1098)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1347)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1099)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1379)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1160)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1280)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1161)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1312)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1162)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1344)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1163)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1376)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1160)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1281)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1161)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1313)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1162)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1345)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1163)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1377)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1160)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1282)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1161)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1314)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1162)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1346)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1163)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1378)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1160)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1283)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1161)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1315)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1162)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1347)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1163)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1379)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1224)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1280)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1225)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1312)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1226)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1344)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1227)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1376)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1224)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1281)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1225)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1313)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1226)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1345)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1227)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1377)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1224)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1282)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1225)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1314)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1226)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1346)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1227)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1378)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1224)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1283)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1225)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1315)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1226)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1347)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1227)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1379)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1288)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1280)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1289)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1312)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1290)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1344)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1291)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1376)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1288)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1281)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1289)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1313)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1290)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1345)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1291)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1377)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1288)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1282)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1289)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1314)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1290)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1346)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1291)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1378)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1288)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1283)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1289)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1315)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1290)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1347)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1291)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1379)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1352)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1280)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1353)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1312)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1354)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1344)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1355)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1376)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1352)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1281)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1353)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1313)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1354)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1345)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1355)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1377)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1352)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1282)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1353)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1314)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1354)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1346)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1355)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1378)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1352)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1283)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1353)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1315)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1354)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1347)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1355)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1379)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1416)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1280)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1417)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1312)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1418)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1344)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1419)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1376)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1416)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1281)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1417)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1313)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1418)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1345)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1419)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1377)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1416)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1282)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1417)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1314)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1418)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1346)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1419)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1378)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1416)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1283)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1417)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1315)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1418)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1347)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1419)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1379)]));
  conv2d_nhwc_local[28] = (conv2d_nhwc_local[28] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1480)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1280)]));
  conv2d_nhwc_local[28] = (conv2d_nhwc_local[28] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1481)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1312)]));
  conv2d_nhwc_local[28] = (conv2d_nhwc_local[28] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1482)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1344)]));
  conv2d_nhwc_local[28] = (conv2d_nhwc_local[28] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1483)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1376)]));
  conv2d_nhwc_local[29] = (conv2d_nhwc_local[29] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1480)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1281)]));
  conv2d_nhwc_local[29] = (conv2d_nhwc_local[29] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1481)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1313)]));
  conv2d_nhwc_local[29] = (conv2d_nhwc_local[29] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1482)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1345)]));
  conv2d_nhwc_local[29] = (conv2d_nhwc_local[29] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1483)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1377)]));
  conv2d_nhwc_local[30] = (conv2d_nhwc_local[30] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1480)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1282)]));
  conv2d_nhwc_local[30] = (conv2d_nhwc_local[30] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1481)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1314)]));
  conv2d_nhwc_local[30] = (conv2d_nhwc_local[30] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1482)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1346)]));
  conv2d_nhwc_local[30] = (conv2d_nhwc_local[30] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1483)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1378)]));
  conv2d_nhwc_local[31] = (conv2d_nhwc_local[31] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1480)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1283)]));
  conv2d_nhwc_local[31] = (conv2d_nhwc_local[31] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1481)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1315)]));
  conv2d_nhwc_local[31] = (conv2d_nhwc_local[31] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1482)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1347)]));
  conv2d_nhwc_local[31] = (conv2d_nhwc_local[31] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1483)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1379)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1036)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1408)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1037)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1440)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1038)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1472)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1039)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1504)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1036)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1409)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1037)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1441)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1038)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1473)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1039)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1505)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1036)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1410)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1037)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1442)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1038)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1474)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1039)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1506)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1036)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1411)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1037)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1443)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1038)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1475)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1039)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1507)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1100)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1408)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1101)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1440)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1102)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1472)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1103)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1504)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1100)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1409)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1101)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1441)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1102)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1473)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1103)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1505)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1100)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1410)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1101)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1442)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1102)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1474)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1103)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1506)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1100)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1411)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1101)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1443)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1102)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1475)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1103)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1507)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1164)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1408)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1165)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1440)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1166)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1472)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1167)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1504)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1164)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1409)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1165)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1441)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1166)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1473)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1167)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1505)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1164)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1410)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1165)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1442)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1166)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1474)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1167)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1506)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1164)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1411)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1165)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1443)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1166)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1475)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1167)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1507)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1228)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1408)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1229)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1440)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1230)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1472)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1231)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1504)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1228)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1409)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1229)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1441)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1230)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1473)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1231)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1505)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1228)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1410)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1229)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1442)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1230)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1474)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1231)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1506)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1228)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1411)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1229)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1443)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1230)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1475)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1231)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1507)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1292)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1408)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1293)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1440)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1294)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1472)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1295)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1504)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1292)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1409)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1293)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1441)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1294)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1473)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1295)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1505)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1292)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1410)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1293)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1442)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1294)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1474)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1295)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1506)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1292)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1411)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1293)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1443)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1294)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1475)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1295)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1507)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1356)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1408)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1357)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1440)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1358)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1472)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1359)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1504)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1356)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1409)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1357)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1441)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1358)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1473)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1359)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1505)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1356)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1410)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1357)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1442)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1358)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1474)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1359)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1506)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1356)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1411)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1357)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1443)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1358)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1475)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1359)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1507)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1420)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1408)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1421)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1440)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1422)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1472)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1423)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1504)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1420)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1409)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1421)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1441)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1422)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1473)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1423)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1505)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1420)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1410)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1421)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1442)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1422)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1474)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1423)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1506)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1420)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1411)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1421)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1443)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1422)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1475)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1423)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1507)]));
  conv2d_nhwc_local[28] = (conv2d_nhwc_local[28] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1484)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1408)]));
  conv2d_nhwc_local[28] = (conv2d_nhwc_local[28] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1485)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1440)]));
  conv2d_nhwc_local[28] = (conv2d_nhwc_local[28] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1486)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1472)]));
  conv2d_nhwc_local[28] = (conv2d_nhwc_local[28] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1487)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1504)]));
  conv2d_nhwc_local[29] = (conv2d_nhwc_local[29] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1484)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1409)]));
  conv2d_nhwc_local[29] = (conv2d_nhwc_local[29] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1485)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1441)]));
  conv2d_nhwc_local[29] = (conv2d_nhwc_local[29] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1486)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1473)]));
  conv2d_nhwc_local[29] = (conv2d_nhwc_local[29] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1487)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1505)]));
  conv2d_nhwc_local[30] = (conv2d_nhwc_local[30] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1484)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1410)]));
  conv2d_nhwc_local[30] = (conv2d_nhwc_local[30] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1485)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1442)]));
  conv2d_nhwc_local[30] = (conv2d_nhwc_local[30] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1486)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1474)]));
  conv2d_nhwc_local[30] = (conv2d_nhwc_local[30] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1487)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1506)]));
  conv2d_nhwc_local[31] = (conv2d_nhwc_local[31] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1484)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1411)]));
  conv2d_nhwc_local[31] = (conv2d_nhwc_local[31] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1485)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1443)]));
  conv2d_nhwc_local[31] = (conv2d_nhwc_local[31] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1486)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1475)]));
  conv2d_nhwc_local[31] = (conv2d_nhwc_local[31] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1487)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1507)]));
__asm__ __volatile__("cp.async.wait_group 0;");

  __syncthreads();
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1536)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1536)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1537)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1568)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1538)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1600)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1539)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1632)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1536)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1537)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1537)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1569)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1538)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1601)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1539)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1633)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1536)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1538)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1537)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1570)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1538)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1602)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1539)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1634)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1536)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1539)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1537)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1571)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1538)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1603)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1539)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1635)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1600)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1536)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1601)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1568)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1602)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1600)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1603)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1632)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1600)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1537)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1601)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1569)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1602)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1601)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1603)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1633)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1600)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1538)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1601)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1570)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1602)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1602)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1603)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1634)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1600)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1539)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1601)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1571)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1602)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1603)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1603)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1635)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1664)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1536)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1665)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1568)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1666)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1600)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1667)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1632)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1664)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1537)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1665)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1569)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1666)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1601)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1667)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1633)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1664)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1538)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1665)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1570)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1666)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1602)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1667)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1634)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1664)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1539)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1665)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1571)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1666)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1603)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1667)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1635)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1728)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1536)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1729)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1568)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1730)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1600)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1731)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1632)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1728)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1537)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1729)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1569)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1730)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1601)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1731)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1633)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1728)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1538)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1729)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1570)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1730)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1602)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1731)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1634)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1728)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1539)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1729)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1571)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1730)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1603)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1731)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1635)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1792)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1536)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1793)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1568)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1794)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1600)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1795)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1632)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1792)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1537)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1793)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1569)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1794)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1601)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1795)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1633)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1792)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1538)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1793)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1570)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1794)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1602)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1795)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1634)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1792)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1539)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1793)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1571)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1794)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1603)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1795)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1635)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1856)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1536)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1857)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1568)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1858)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1600)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1859)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1632)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1856)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1537)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1857)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1569)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1858)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1601)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1859)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1633)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1856)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1538)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1857)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1570)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1858)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1602)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1859)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1634)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1856)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1539)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1857)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1571)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1858)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1603)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1859)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1635)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1920)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1536)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1921)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1568)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1922)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1600)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1923)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1632)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1920)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1537)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1921)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1569)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1922)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1601)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1923)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1633)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1920)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1538)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1921)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1570)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1922)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1602)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1923)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1634)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1920)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1539)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1921)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1571)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1922)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1603)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1923)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1635)]));
  conv2d_nhwc_local[28] = (conv2d_nhwc_local[28] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1984)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1536)]));
  conv2d_nhwc_local[28] = (conv2d_nhwc_local[28] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1985)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1568)]));
  conv2d_nhwc_local[28] = (conv2d_nhwc_local[28] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1986)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1600)]));
  conv2d_nhwc_local[28] = (conv2d_nhwc_local[28] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1987)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1632)]));
  conv2d_nhwc_local[29] = (conv2d_nhwc_local[29] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1984)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1537)]));
  conv2d_nhwc_local[29] = (conv2d_nhwc_local[29] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1985)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1569)]));
  conv2d_nhwc_local[29] = (conv2d_nhwc_local[29] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1986)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1601)]));
  conv2d_nhwc_local[29] = (conv2d_nhwc_local[29] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1987)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1633)]));
  conv2d_nhwc_local[30] = (conv2d_nhwc_local[30] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1984)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1538)]));
  conv2d_nhwc_local[30] = (conv2d_nhwc_local[30] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1985)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1570)]));
  conv2d_nhwc_local[30] = (conv2d_nhwc_local[30] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1986)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1602)]));
  conv2d_nhwc_local[30] = (conv2d_nhwc_local[30] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1987)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1634)]));
  conv2d_nhwc_local[31] = (conv2d_nhwc_local[31] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1984)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1539)]));
  conv2d_nhwc_local[31] = (conv2d_nhwc_local[31] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1985)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1571)]));
  conv2d_nhwc_local[31] = (conv2d_nhwc_local[31] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1986)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1603)]));
  conv2d_nhwc_local[31] = (conv2d_nhwc_local[31] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1987)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1635)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1540)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1664)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1541)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1696)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1542)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1728)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1543)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1760)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1540)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1665)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1541)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1697)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1542)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1729)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1543)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1761)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1540)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1666)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1541)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1698)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1542)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1730)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1543)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1762)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1540)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1667)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1541)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1699)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1542)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1731)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1543)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1763)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1604)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1664)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1605)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1696)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1606)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1728)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1607)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1760)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1604)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1665)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1605)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1697)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1606)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1729)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1607)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1761)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1604)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1666)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1605)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1698)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1606)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1730)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1607)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1762)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1604)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1667)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1605)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1699)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1606)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1731)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1607)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1763)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1668)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1664)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1669)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1696)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1670)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1728)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1671)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1760)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1668)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1665)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1669)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1697)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1670)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1729)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1671)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1761)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1668)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1666)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1669)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1698)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1670)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1730)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1671)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1762)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1668)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1667)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1669)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1699)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1670)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1731)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1671)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1763)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1732)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1664)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1733)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1696)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1734)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1728)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1735)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1760)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1732)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1665)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1733)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1697)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1734)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1729)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1735)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1761)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1732)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1666)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1733)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1698)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1734)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1730)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1735)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1762)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1732)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1667)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1733)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1699)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1734)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1731)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1735)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1763)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1796)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1664)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1797)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1696)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1798)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1728)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1799)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1760)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1796)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1665)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1797)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1697)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1798)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1729)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1799)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1761)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1796)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1666)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1797)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1698)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1798)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1730)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1799)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1762)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1796)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1667)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1797)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1699)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1798)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1731)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1799)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1763)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1860)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1664)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1861)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1696)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1862)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1728)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1863)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1760)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1860)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1665)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1861)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1697)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1862)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1729)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1863)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1761)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1860)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1666)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1861)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1698)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1862)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1730)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1863)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1762)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1860)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1667)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1861)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1699)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1862)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1731)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1863)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1763)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1924)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1664)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1925)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1696)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1926)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1728)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1927)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1760)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1924)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1665)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1925)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1697)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1926)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1729)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1927)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1761)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1924)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1666)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1925)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1698)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1926)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1730)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1927)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1762)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1924)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1667)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1925)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1699)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1926)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1731)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1927)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1763)]));
  conv2d_nhwc_local[28] = (conv2d_nhwc_local[28] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1988)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1664)]));
  conv2d_nhwc_local[28] = (conv2d_nhwc_local[28] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1989)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1696)]));
  conv2d_nhwc_local[28] = (conv2d_nhwc_local[28] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1990)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1728)]));
  conv2d_nhwc_local[28] = (conv2d_nhwc_local[28] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1991)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1760)]));
  conv2d_nhwc_local[29] = (conv2d_nhwc_local[29] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1988)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1665)]));
  conv2d_nhwc_local[29] = (conv2d_nhwc_local[29] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1989)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1697)]));
  conv2d_nhwc_local[29] = (conv2d_nhwc_local[29] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1990)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1729)]));
  conv2d_nhwc_local[29] = (conv2d_nhwc_local[29] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1991)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1761)]));
  conv2d_nhwc_local[30] = (conv2d_nhwc_local[30] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1988)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1666)]));
  conv2d_nhwc_local[30] = (conv2d_nhwc_local[30] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1989)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1698)]));
  conv2d_nhwc_local[30] = (conv2d_nhwc_local[30] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1990)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1730)]));
  conv2d_nhwc_local[30] = (conv2d_nhwc_local[30] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1991)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1762)]));
  conv2d_nhwc_local[31] = (conv2d_nhwc_local[31] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1988)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1667)]));
  conv2d_nhwc_local[31] = (conv2d_nhwc_local[31] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1989)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1699)]));
  conv2d_nhwc_local[31] = (conv2d_nhwc_local[31] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1990)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1731)]));
  conv2d_nhwc_local[31] = (conv2d_nhwc_local[31] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1991)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1763)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1544)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1792)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1545)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1824)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1546)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1856)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1547)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1888)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1544)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1793)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1545)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1825)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1546)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1857)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1547)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1889)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1544)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1794)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1545)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1826)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1546)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1858)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1547)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1890)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1544)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1795)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1545)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1827)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1546)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1859)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1547)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1891)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1608)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1792)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1609)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1824)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1610)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1856)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1611)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1888)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1608)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1793)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1609)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1825)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1610)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1857)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1611)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1889)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1608)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1794)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1609)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1826)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1610)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1858)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1611)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1890)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1608)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1795)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1609)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1827)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1610)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1859)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1611)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1891)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1672)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1792)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1673)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1824)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1674)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1856)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1675)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1888)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1672)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1793)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1673)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1825)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1674)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1857)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1675)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1889)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1672)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1794)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1673)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1826)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1674)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1858)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1675)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1890)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1672)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1795)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1673)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1827)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1674)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1859)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1675)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1891)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1736)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1792)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1737)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1824)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1738)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1856)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1739)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1888)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1736)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1793)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1737)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1825)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1738)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1857)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1739)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1889)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1736)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1794)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1737)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1826)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1738)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1858)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1739)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1890)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1736)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1795)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1737)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1827)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1738)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1859)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1739)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1891)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1800)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1792)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1801)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1824)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1802)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1856)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1803)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1888)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1800)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1793)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1801)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1825)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1802)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1857)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1803)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1889)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1800)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1794)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1801)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1826)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1802)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1858)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1803)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1890)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1800)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1795)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1801)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1827)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1802)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1859)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1803)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1891)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1864)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1792)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1865)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1824)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1866)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1856)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1867)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1888)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1864)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1793)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1865)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1825)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1866)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1857)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1867)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1889)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1864)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1794)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1865)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1826)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1866)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1858)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1867)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1890)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1864)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1795)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1865)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1827)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1866)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1859)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1867)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1891)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1928)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1792)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1929)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1824)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1930)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1856)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1931)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1888)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1928)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1793)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1929)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1825)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1930)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1857)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1931)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1889)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1928)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1794)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1929)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1826)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1930)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1858)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1931)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1890)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1928)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1795)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1929)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1827)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1930)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1859)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1931)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1891)]));
  conv2d_nhwc_local[28] = (conv2d_nhwc_local[28] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1992)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1792)]));
  conv2d_nhwc_local[28] = (conv2d_nhwc_local[28] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1993)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1824)]));
  conv2d_nhwc_local[28] = (conv2d_nhwc_local[28] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1994)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1856)]));
  conv2d_nhwc_local[28] = (conv2d_nhwc_local[28] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1995)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1888)]));
  conv2d_nhwc_local[29] = (conv2d_nhwc_local[29] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1992)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1793)]));
  conv2d_nhwc_local[29] = (conv2d_nhwc_local[29] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1993)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1825)]));
  conv2d_nhwc_local[29] = (conv2d_nhwc_local[29] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1994)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1857)]));
  conv2d_nhwc_local[29] = (conv2d_nhwc_local[29] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1995)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1889)]));
  conv2d_nhwc_local[30] = (conv2d_nhwc_local[30] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1992)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1794)]));
  conv2d_nhwc_local[30] = (conv2d_nhwc_local[30] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1993)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1826)]));
  conv2d_nhwc_local[30] = (conv2d_nhwc_local[30] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1994)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1858)]));
  conv2d_nhwc_local[30] = (conv2d_nhwc_local[30] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1995)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1890)]));
  conv2d_nhwc_local[31] = (conv2d_nhwc_local[31] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1992)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1795)]));
  conv2d_nhwc_local[31] = (conv2d_nhwc_local[31] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1993)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1827)]));
  conv2d_nhwc_local[31] = (conv2d_nhwc_local[31] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1994)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1859)]));
  conv2d_nhwc_local[31] = (conv2d_nhwc_local[31] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1995)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1891)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1548)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1920)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1549)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1952)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1550)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1984)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1551)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 2016)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1548)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1921)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1549)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1953)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1550)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1985)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1551)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 2017)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1548)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1922)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1549)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1954)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1550)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1986)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1551)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 2018)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1548)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1923)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1549)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1955)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1550)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1987)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1551)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 2019)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1612)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1920)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1613)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1952)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1614)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1984)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1615)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 2016)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1612)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1921)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1613)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1953)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1614)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1985)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1615)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 2017)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1612)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1922)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1613)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1954)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1614)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1986)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1615)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 2018)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1612)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1923)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1613)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1955)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1614)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1987)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1615)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 2019)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1676)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1920)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1677)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1952)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1678)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1984)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1679)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 2016)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1676)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1921)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1677)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1953)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1678)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1985)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1679)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 2017)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1676)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1922)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1677)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1954)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1678)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1986)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1679)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 2018)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1676)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1923)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1677)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1955)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1678)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1987)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1679)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 2019)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1740)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1920)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1741)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1952)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1742)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1984)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1743)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 2016)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1740)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1921)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1741)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1953)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1742)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1985)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1743)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 2017)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1740)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1922)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1741)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1954)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1742)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1986)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1743)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 2018)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1740)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1923)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1741)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1955)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1742)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1987)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1743)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 2019)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1804)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1920)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1805)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1952)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1806)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1984)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1807)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 2016)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1804)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1921)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1805)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1953)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1806)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1985)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1807)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 2017)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1804)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1922)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1805)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1954)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1806)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1986)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1807)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 2018)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1804)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1923)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1805)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1955)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1806)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1987)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1807)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 2019)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1868)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1920)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1869)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1952)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1870)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1984)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1871)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 2016)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1868)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1921)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1869)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1953)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1870)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1985)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1871)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 2017)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1868)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1922)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1869)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1954)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1870)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1986)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1871)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 2018)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1868)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1923)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1869)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1955)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1870)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1987)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1871)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 2019)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1932)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1920)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1933)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1952)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1934)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1984)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1935)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 2016)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1932)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1921)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1933)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1953)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1934)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1985)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1935)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 2017)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1932)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1922)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1933)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1954)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1934)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1986)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1935)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 2018)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1932)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1923)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1933)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1955)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1934)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1987)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1935)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 2019)]));
  conv2d_nhwc_local[28] = (conv2d_nhwc_local[28] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1996)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1920)]));
  conv2d_nhwc_local[28] = (conv2d_nhwc_local[28] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1997)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1952)]));
  conv2d_nhwc_local[28] = (conv2d_nhwc_local[28] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1998)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1984)]));
  conv2d_nhwc_local[28] = (conv2d_nhwc_local[28] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1999)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 2016)]));
  conv2d_nhwc_local[29] = (conv2d_nhwc_local[29] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1996)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1921)]));
  conv2d_nhwc_local[29] = (conv2d_nhwc_local[29] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1997)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1953)]));
  conv2d_nhwc_local[29] = (conv2d_nhwc_local[29] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1998)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1985)]));
  conv2d_nhwc_local[29] = (conv2d_nhwc_local[29] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1999)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 2017)]));
  conv2d_nhwc_local[30] = (conv2d_nhwc_local[30] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1996)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1922)]));
  conv2d_nhwc_local[30] = (conv2d_nhwc_local[30] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1997)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1954)]));
  conv2d_nhwc_local[30] = (conv2d_nhwc_local[30] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1998)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1986)]));
  conv2d_nhwc_local[30] = (conv2d_nhwc_local[30] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1999)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 2018)]));
  conv2d_nhwc_local[31] = (conv2d_nhwc_local[31] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1996)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1923)]));
  conv2d_nhwc_local[31] = (conv2d_nhwc_local[31] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1997)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1955)]));
  conv2d_nhwc_local[31] = (conv2d_nhwc_local[31] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1998)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1987)]));
  conv2d_nhwc_local[31] = (conv2d_nhwc_local[31] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1999)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 2019)]));
  conv2d_nhwc[((((((((int)blockIdx.x) / 28) * 28672) + (((((int)blockIdx.x) % 28) >> 1) * 256)) + ((((int)threadIdx.x) >> 3) * 64)) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 7) * 4))] = conv2d_nhwc_local[0];
  conv2d_nhwc[(((((((((int)blockIdx.x) / 28) * 28672) + (((((int)blockIdx.x) % 28) >> 1) * 256)) + ((((int)threadIdx.x) >> 3) * 64)) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 1)] = conv2d_nhwc_local[1];
  conv2d_nhwc[(((((((((int)blockIdx.x) / 28) * 28672) + (((((int)blockIdx.x) % 28) >> 1) * 256)) + ((((int)threadIdx.x) >> 3) * 64)) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 2)] = conv2d_nhwc_local[2];
  conv2d_nhwc[(((((((((int)blockIdx.x) / 28) * 28672) + (((((int)blockIdx.x) % 28) >> 1) * 256)) + ((((int)threadIdx.x) >> 3) * 64)) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 3)] = conv2d_nhwc_local[3];
  conv2d_nhwc[(((((((((int)blockIdx.x) / 28) * 28672) + (((((int)blockIdx.x) % 28) >> 1) * 256)) + ((((int)threadIdx.x) >> 3) * 64)) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 3584)] = conv2d_nhwc_local[4];
  conv2d_nhwc[(((((((((int)blockIdx.x) / 28) * 28672) + (((((int)blockIdx.x) % 28) >> 1) * 256)) + ((((int)threadIdx.x) >> 3) * 64)) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 3585)] = conv2d_nhwc_local[5];
  conv2d_nhwc[(((((((((int)blockIdx.x) / 28) * 28672) + (((((int)blockIdx.x) % 28) >> 1) * 256)) + ((((int)threadIdx.x) >> 3) * 64)) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 3586)] = conv2d_nhwc_local[6];
  conv2d_nhwc[(((((((((int)blockIdx.x) / 28) * 28672) + (((((int)blockIdx.x) % 28) >> 1) * 256)) + ((((int)threadIdx.x) >> 3) * 64)) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 3587)] = conv2d_nhwc_local[7];
  conv2d_nhwc[(((((((((int)blockIdx.x) / 28) * 28672) + (((((int)blockIdx.x) % 28) >> 1) * 256)) + ((((int)threadIdx.x) >> 3) * 64)) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 7168)] = conv2d_nhwc_local[8];
  conv2d_nhwc[(((((((((int)blockIdx.x) / 28) * 28672) + (((((int)blockIdx.x) % 28) >> 1) * 256)) + ((((int)threadIdx.x) >> 3) * 64)) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 7169)] = conv2d_nhwc_local[9];
  conv2d_nhwc[(((((((((int)blockIdx.x) / 28) * 28672) + (((((int)blockIdx.x) % 28) >> 1) * 256)) + ((((int)threadIdx.x) >> 3) * 64)) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 7170)] = conv2d_nhwc_local[10];
  conv2d_nhwc[(((((((((int)blockIdx.x) / 28) * 28672) + (((((int)blockIdx.x) % 28) >> 1) * 256)) + ((((int)threadIdx.x) >> 3) * 64)) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 7171)] = conv2d_nhwc_local[11];
  conv2d_nhwc[(((((((((int)blockIdx.x) / 28) * 28672) + (((((int)blockIdx.x) % 28) >> 1) * 256)) + ((((int)threadIdx.x) >> 3) * 64)) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 10752)] = conv2d_nhwc_local[12];
  conv2d_nhwc[(((((((((int)blockIdx.x) / 28) * 28672) + (((((int)blockIdx.x) % 28) >> 1) * 256)) + ((((int)threadIdx.x) >> 3) * 64)) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 10753)] = conv2d_nhwc_local[13];
  conv2d_nhwc[(((((((((int)blockIdx.x) / 28) * 28672) + (((((int)blockIdx.x) % 28) >> 1) * 256)) + ((((int)threadIdx.x) >> 3) * 64)) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 10754)] = conv2d_nhwc_local[14];
  conv2d_nhwc[(((((((((int)blockIdx.x) / 28) * 28672) + (((((int)blockIdx.x) % 28) >> 1) * 256)) + ((((int)threadIdx.x) >> 3) * 64)) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 10755)] = conv2d_nhwc_local[15];
  conv2d_nhwc[(((((((((int)blockIdx.x) / 28) * 28672) + (((((int)blockIdx.x) % 28) >> 1) * 256)) + ((((int)threadIdx.x) >> 3) * 64)) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 14336)] = conv2d_nhwc_local[16];
  conv2d_nhwc[(((((((((int)blockIdx.x) / 28) * 28672) + (((((int)blockIdx.x) % 28) >> 1) * 256)) + ((((int)threadIdx.x) >> 3) * 64)) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 14337)] = conv2d_nhwc_local[17];
  conv2d_nhwc[(((((((((int)blockIdx.x) / 28) * 28672) + (((((int)blockIdx.x) % 28) >> 1) * 256)) + ((((int)threadIdx.x) >> 3) * 64)) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 14338)] = conv2d_nhwc_local[18];
  conv2d_nhwc[(((((((((int)blockIdx.x) / 28) * 28672) + (((((int)blockIdx.x) % 28) >> 1) * 256)) + ((((int)threadIdx.x) >> 3) * 64)) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 14339)] = conv2d_nhwc_local[19];
  conv2d_nhwc[(((((((((int)blockIdx.x) / 28) * 28672) + (((((int)blockIdx.x) % 28) >> 1) * 256)) + ((((int)threadIdx.x) >> 3) * 64)) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 17920)] = conv2d_nhwc_local[20];
  conv2d_nhwc[(((((((((int)blockIdx.x) / 28) * 28672) + (((((int)blockIdx.x) % 28) >> 1) * 256)) + ((((int)threadIdx.x) >> 3) * 64)) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 17921)] = conv2d_nhwc_local[21];
  conv2d_nhwc[(((((((((int)blockIdx.x) / 28) * 28672) + (((((int)blockIdx.x) % 28) >> 1) * 256)) + ((((int)threadIdx.x) >> 3) * 64)) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 17922)] = conv2d_nhwc_local[22];
  conv2d_nhwc[(((((((((int)blockIdx.x) / 28) * 28672) + (((((int)blockIdx.x) % 28) >> 1) * 256)) + ((((int)threadIdx.x) >> 3) * 64)) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 17923)] = conv2d_nhwc_local[23];
  conv2d_nhwc[(((((((((int)blockIdx.x) / 28) * 28672) + (((((int)blockIdx.x) % 28) >> 1) * 256)) + ((((int)threadIdx.x) >> 3) * 64)) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 21504)] = conv2d_nhwc_local[24];
  conv2d_nhwc[(((((((((int)blockIdx.x) / 28) * 28672) + (((((int)blockIdx.x) % 28) >> 1) * 256)) + ((((int)threadIdx.x) >> 3) * 64)) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 21505)] = conv2d_nhwc_local[25];
  conv2d_nhwc[(((((((((int)blockIdx.x) / 28) * 28672) + (((((int)blockIdx.x) % 28) >> 1) * 256)) + ((((int)threadIdx.x) >> 3) * 64)) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 21506)] = conv2d_nhwc_local[26];
  conv2d_nhwc[(((((((((int)blockIdx.x) / 28) * 28672) + (((((int)blockIdx.x) % 28) >> 1) * 256)) + ((((int)threadIdx.x) >> 3) * 64)) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 21507)] = conv2d_nhwc_local[27];
  conv2d_nhwc[(((((((((int)blockIdx.x) / 28) * 28672) + (((((int)blockIdx.x) % 28) >> 1) * 256)) + ((((int)threadIdx.x) >> 3) * 64)) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 25088)] = conv2d_nhwc_local[28];
  conv2d_nhwc[(((((((((int)blockIdx.x) / 28) * 28672) + (((((int)blockIdx.x) % 28) >> 1) * 256)) + ((((int)threadIdx.x) >> 3) * 64)) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 25089)] = conv2d_nhwc_local[29];
  conv2d_nhwc[(((((((((int)blockIdx.x) / 28) * 28672) + (((((int)blockIdx.x) % 28) >> 1) * 256)) + ((((int)threadIdx.x) >> 3) * 64)) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 25090)] = conv2d_nhwc_local[30];
  conv2d_nhwc[(((((((((int)blockIdx.x) / 28) * 28672) + (((((int)blockIdx.x) % 28) >> 1) * 256)) + ((((int)threadIdx.x) >> 3) * 64)) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 25091)] = conv2d_nhwc_local[31];
}


